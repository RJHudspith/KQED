#include "hip/hip_runtime.h"
#include "KQED.h"      // definitions and enums

#include "cheby.h"     // chebUsum and alike
#include "getff-new.h" // alphabetising

// interpolation function
// dy should be set to y1-y2
__device__
static inline double
interpol3( const double y ,
	   const double y1 ,
	   const double y2 ,
	   const double dy ,
	   const double f1 ,
	   const double f2 ,
	   const double g1 ,
	   const double g2 )
{
  return ((-f1* (y - y2)*(y - y2)*(2* y - 3* y1 + y2) +			\
	   (y - y1)* (f2* (y - y1)*(2* y + y1 - 3* y2)			\
		      + (y - y2)*dy* (g1* y + g2* y - g2* y1 - g1* y2)))/(dy*dy*dy)) ;
}

// precompute all this business for x or y
__device__ KQED_PRIVATE
void
precompute_INV( struct intprecomp *INVy ,
		const double y ,
		const double y1 ,
		const double y2 ,
		const size_t idx )
{
  INVy -> idx = idx ;
  register const double dy = y1-y2 ;
  register const double ymy1 = y-y1 ;
  register const double ymy2 = y-y2 ;
  register const double ym2sq = (ymy2*ymy2) ;
  register const double ym1sq = (ymy1*ymy1) ;
  INVy -> A = -(ym2sq)*(2*y - 3*y1 + y2) ;
  INVy -> B = (ym1sq)*(2*y + y1 - 3*y2) ;
  INVy -> C1 = (ymy1)*(ym2sq)*dy ;
  INVy -> C2 = (ym1sq)*(ymy2)*dy ;
  INVy -> D = 1./(dy*dy*dy) ;
  INVy -> lA = (y2-y)/(y2-y1) ;
}

// precompute all this business for x
__device__ KQED_PRIVATE
void
precompute_INVx( struct intprecomp *INVx ,
    const double y ,
    const double y1 ,
    const double y2 ,
    const size_t idx )
{
  precompute_INV(INVx, y, y1, y2, idx);
}

__device__
static inline double
interpol4( const struct intprecomp INVx ,
	   const double f1 ,
	   const double f2 ,
	   const double g1 ,
	   const double g2 )
{
  return (f1*INVx.A + f2*INVx.B + g1*INVx.C1 + g2*INVx.C2)*INVx.D ; 
}

// function pointer for cheby stuff
// static double (*Func_usm[4])( const int , const double , const double *) =
// { chebUsum , dchebUsum , ddchebUsum , dddchebUsum } ;
__device__
static double Func_usm( const int i,
    const int n, const double x, const double *f ) {
  switch(i) {
    case 0: return chebUsum(n, x, f);
    case 1: return dchebUsum(n, x, f);
    case 2: return ddchebUsum(n, x, f);
    case 3: return dddchebUsum(n, x, f);
    default: return 0.0;
  }
}

// returns the form factor, given the coefficients fm[0..(nf-1)] and fp[0..(nf-1)]
// e.g. fm = alpha^{(3)}_{m-}  and fp = alpha^{(3)}_{m+}
// nf = length of vectors fm[nm][ix/iy] and similarly for fp
// nm = index of the sum sigma that appeared in the integrand == outer index of ff
// ndy and ndcb = # derivatives with respect to y and cos(beta) respectively
// x is not used in this function
__device__
static void
getff2( double res[2] ,
	const int nf, const FFidx nm ,
	const bool ndy , const NDCB ndcb, 
	const double y, const double x ,
	const float *fm, const float *fp )
{
  const double y1 = 1.0/y;

  // enum guarantees these are set but avoid stupid
  // gcc maybe unitialized warning
  double yp = 1 ;
  int mshm = 2 ;
  switch(nm) {
  case QG0    : case dxQG0 : yp = 1.0   ; mshm=2 ; break;
  case QG1    : case dxQG1 : yp = 1.0   ; mshm=2 ; break;
  case QG2    : case dxQG2 : case d2xQG2: yp =  y1 ; mshm=3 ; break;
  case QG3    : case dxQG3 : case d2xQG3: yp = 1.0 ; mshm=2 ; break;
  case QL4    : case dxQL4 : yp =   y   ; mshm=1 ; break;
  case QL2    : case dxQL2 : yp = y1*y1 ; mshm=4 ; break;
  }

  // these two parameters are simply related to mshm
  int mshp = 2 - mshm ;
  // mm used to be set by the global map Idm, but that was unnecessary
  const int mm = abs( mshp ) ;

  // set fval to zero, actually not really needed
  double fval[128], fvalD[128];
  // TODO: do something more reasonable if too long
  if (nf+mm >= 128) return;
  memset( fval , 0 , (nf+mm)*sizeof( double ) ) ;
  memset( fvalD , 0 , (nf+mm)*sizeof( double ) ) ;
  double facm = y1*y1*yp ;
  double facp = yp ;

  int j ;
  for( j = 0 ; j < mm ; j++ ) {
    facm *= y1 ;
    facp *= y ;
  }

  double *Pfval = (double*)fval + mm ;
  double *PfvalD = (double*)fvalD + mm ;

  mshm += mm ; mshp += mm ;
  for(j=0;j<nf;j++) {
    *PfvalD = y1*mshp*facp*(*fp) - y1*mshm*facm*(*fm) ;
    *Pfval = facm*(*fm) + facp*(*fp) ;
    facm *= y1; facp *= y;    
    Pfval++ ; PfvalD++ ; fm++ ; fp++ ; mshm++ ; mshp++ ;
  }

  res[1] = Func_usm(mm + ndcb, nf+mm, x , fvalD ) ;
  res[0] = ndy? res[1] : Func_usm(mm + ndcb, nf+mm, x , fval ) ;
}

// case where you have read in the weight functions upon initialization
// interpolates the form factor[nm] to the target point y using the grid
__device__ KQED_PRIVATE
double
accessv( const bool flag_hy, const bool use_y_derivs,
	 const int ix, const int iy,
	 const FFidx nm, const bool ndy, const NDCB ndcb,
	 const double cb, const double y, 
	 const struct Grid_coeffs Grid )
{
  const int nx = Grid.nfx[ix] ;
  const double y1 = Grid.YY[iy];
  double res1[2] = {0.,0.} ;
    
  // occasionally ndy gets set to 1, if use_y_derivs is set we always do the deriv
  getff2( res1 , nx, nm, ndy, ndcb, y1, cb,
	  getFfm(&Grid, nm, ix, iy) , getFfp(&Grid, nm, ix, iy) ) ;
  
  // if we are not at the upper limit of Y we can use info from the next point
  if(!flag_hy) {
    const int iy2 = iy+1;
    const double y2 = Grid.YY[iy2];
    double res2[2] = {0.,0.} ;

    getff2( res2 , nx, nm, ndy, ndcb, y2, cb,
	    getFfm(&Grid, nm, ix, iy2) ,
	    getFfp(&Grid, nm, ix, iy2) ) ;
    
    if( use_y_derivs ) {
      return interpol3( y, y1, y2, y1-y2,
			res1[0], res2[0], res1[1], res2[1] ) ;      
    } else {
      // value interpolated to target y, at x=x1[0];
      return lerp( (y2-y)/(y2-y1) , res1[0] , res2[0] ) ;
    }
  }
  return res1[0] ;
}

// returns the lower index that bounds "target"
// e.g arr[lo] < target < arr[lo+1]
// (assumes a monotonically increasing arr)
__device__
static int
bsrch( const double *arr , const double target ,
  const int lo , const int hi )
{
  // when hi == lo we are done
  if( ( hi - lo ) < 2 ) return lo ;
  const int mid = ( hi + lo )/2 ;
  if( arr[mid] > target ) {
    return bsrch( arr , target , lo , mid ) ;
  } else {
    return bsrch( arr , target , mid , hi ) ;
  }
}

// linear search variant targeted at avoiding warp divergence and keeping
// regular memory access on CUDA.
__device__
static int
lsrch( const double *arr, const double target,
    const int lo, const int hi ) {
  int index = lo;
  for (int i = lo; i < hi; ++i) {
    index = (arr[i] <= target && target <= arr[i+1]) ? i : index;
  }
  return index;
}

// returns the lower index that bounds "target"
// e.g arr[lo] < target < arr[lo+1]
// (assumes a monotonically increasing arr)
__device__ KQED_PRIVATE
int
find_ind(const double *arr, const double target,
    const int lo, const int hi) {
  // NOTE: this is not well-suited for GPU execution
  // return bsrch(arr, target, lo, hi);
  // Better option:
  return lsrch(arr, target, lo, hi);
}

// extract the form factor
__device__ KQED_PRIVATE
double
extractff( const FFidx nm, const bool ndy, const NDCB ndcb,
	   const struct invariants Inv , const struct Grid_coeffs Grid )
{  
  const bool flag_hx = ( Inv.x >= Grid.XX[ Grid.nstpx-1 ] ) ;
  const bool flag_hy = ( Inv.y >= Grid.YY[ Grid.nstpy-1 ] ) ;
 
  const bool use_x_derivs = (nm<dxQG0 || nm==dxQG2 || nm==dxQG3) ;

  const int ix = Inv.INVx.idx , iy = Inv.INVy.idx ; 
  const bool use_y_derivs = (ndy==false) ;
  const double f1iy = accessv( flag_hy, use_y_derivs, ix, iy, nm,
			       ndy, ndcb, Inv.cb, Inv.y, Grid ) ;

  if(!flag_hx) {
    const double f2iy = accessv( flag_hy, use_y_derivs, ix+1, iy, nm,
				 ndy, ndcb, Inv.cb, Inv.y , Grid ) ;
    if(use_x_derivs) {
      const int offset = nm < dxQG0 ? dxQG0 : QL4 ;
      // These enum additions are pretty sketchy...
      const double g1iy = accessv( flag_hy, use_y_derivs, ix, iy, (FFidx)(nm+offset),
				   ndy, ndcb, Inv.cb, Inv.y, Grid ) ;
      const double g2iy = accessv( flag_hy, use_y_derivs, ix+1, iy, (FFidx)(nm+offset),
				   ndy, ndcb, Inv.cb, Inv.y, Grid ) ;

      return interpol4( Inv.INVx, f1iy, f2iy, g1iy, g2iy ) ;
    } else {
      // lerpity lerp
      return lerp( Inv.INVx.lA , f1iy , f2iy ) ;
    }
  }
  return f1iy ;
}

// extract the form factor
__device__ KQED_PRIVATE
void
extractff2( const FFidx nm,
	    const NDCB ndcb,
	    const struct invariants Inv ,
	    const struct Grid_coeffs Grid ,
	    double F[4] )
{  
  // derivative map
  const NDCB ndcb2 = (NDCB)(ndcb+1 < 5 ? ndcb+1 : ndcb) ;
  // map for the x-derivative, incomplete as some derivatives aren't used
  const FFidx dxmap[14] = { dxQG0  , dxQG1 , dxQG2  , dxQG3  , dxQL4 , dxQL2 ,
			  dxQG0  , dxQG1 , d2xQG2 , d2xQG3 , dxQL4 , dxQL2 ,
			  d2xQG2 , d2xQG3 } ;
  // derivative wrt dcb
  F[0] = extractff( nm, false , ndcb2 , Inv , Grid ) ;
  // derivative wrt x
  F[1] = extractff( dxmap[nm], false , ndcb  , Inv , Grid ) ;
  // derivative wrt y
  F[2] = extractff( nm, true  , ndcb  , Inv , Grid ) ;
  // no derivative
  F[3] = extractff( nm, false , ndcb  , Inv , Grid ) ;
  return ;
}

// Initialises the invariants used in chnr_*
__device__ KQED_PRIVATE
struct invariants
set_invariants( const double xv[4] ,
		const double yv[4] ,
		const struct Grid_coeffs Grid )
{
  struct invariants Inv ;
  const double EPSIN = 1E-7 ;
  
  Inv.xsq = SCALPROD( xv , xv ) ;
  Inv.xsq = Inv.xsq < EPSIN ? EPSIN : Inv.xsq ;
  Inv.x = sqrt( fabs( Inv.xsq ) ) ;

  // y needs a little fudge factor as it is badly behaved for very small y
  Inv.ysq = SCALPROD( yv , yv ) ;
  Inv.ysq = Inv.ysq < EPSIN ? EPSIN : Inv.ysq ;
  Inv.y = sqrt( fabs( Inv.ysq ) ) ;

  Inv.xdoty = SCALPROD( xv , yv ) ;
  Inv.cb = Inv.xdoty/( Inv.x * Inv.y ) ;

  Inv.cborig = Inv.cb ;
  Inv.yorig = Inv.y ;
  Inv.xmysq = ( Inv.xsq + Inv.ysq )*(1.00000000000001)-2.0*Inv.xdoty ;
  Inv.xmy = sqrt( fabs( Inv.xmysq ) ) ;

  Inv.flag2 = false ;
  const double rx = ( Inv.x > Inv.xmy ? Inv.xmy/Inv.x : Inv.x/Inv.xmy);
  const double rxy = ( Inv.x > Inv.y ? Inv.y/Inv.x : Inv.x/Inv.y);

  if( rx < rxy
      && Inv.xmy < Grid.YY[ Grid.nstpy-1] 
      && fabs(Inv.xmysq) > 1E-28 ) {
    Inv.cb = (Inv.x-Inv.y*Inv.cb)/Inv.xmy ;
    Inv.y = Inv.xmy ;    
    Inv.flag2 = true ;
  }

  // setup INVx
  const size_t ix1 = (size_t)find_ind( Grid.XX , Inv.x , 0 , Grid.nstpx ) ;
  size_t ix2 = ix1+1 ;
  if( ix2 >= (size_t)Grid.nstpx ) {
    ix2 = ix1 ;
  }
  precompute_INVx( &Inv.INVx, Inv.x, Grid.XX[ix1], Grid.XX[ix2] , ix1 ) ;
  
  // setup InvY
  const size_t iy1 = (size_t)find_ind( Grid.YY , Inv.y , 0 , Grid.nstpy ) ;  
  // y edge case
  size_t iy2 = iy1 + 1 ;
  if( iy2 >= (size_t)Grid.nstpy ) {
    iy2 = iy1 ;
  }
  precompute_INV( &Inv.INVy , Inv.y , Grid.YY[iy1] , Grid.YY[iy2] , iy1 ) ;
  
  return Inv ;
}
