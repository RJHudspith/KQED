/**
   @file io.c
   @brief reading and writing stuff
 */
#include "KQED.h"

#include "crc32c.h"      // DML_checksum_accum
#include "GLU_bswap.h"   // byte swaps if we are different endian to file format

#define FILE_FF "/PRECOMP/FFxy_single_cksum.bin"
#define FILE_TAYLORX "/PRECOMP/taylorx_cksum.bin"
#define FILE_TAYLORY "/PRECOMP/taylory_cksum.bin" 

__host__
static int
FREAD32( void *p , const size_t size , const size_t length , FILE *f )
{
  if( fread( p , size , length , f ) != length ) {
    fprintf( stderr , "FREAD32 failure\n" ) ;
    return 1 ;
  }
#ifdef WORDS_BIGENDIAN
  bswap_32( length , p ) ;
#endif
  return 0 ;
}

__host__
static int
FREAD64( void *p , const size_t size , const size_t length , FILE *f )
{
  if( fread( p , size , length , f ) != length ) {
    fprintf( stderr , "FREAD32 failure\n" ) ;
    return 1 ;
  }
#ifdef WORDS_BIGENDIAN
  bswap_32( length , p ) ;
#endif
  return 0 ;
}

// read the (single-precision) binary file allocating Grid objects
__host__
int
read_ff( struct Grid_coeffs *Grid )
{
  char filestr[ strlen( LIB_PATH ) + strlen( FILE_FF ) + 1 ] ;
  sprintf( filestr , "%s%s" , LIB_PATH , FILE_FF ) ;
  FILE *fr = fopen( filestr , "rb" ) ;

  if( fr == NULL ) {
    fprintf( stderr , "Cannot find %s\n" , filestr ) ;
    return 1 ;
  }

  int moniker ;
  FREAD32( &moniker , sizeof( int ) , 1 , fr ) ;

  if( moniker != 816968 ) {
    fprintf( stderr , "|IO| missing magic number in FFxy_single.bin\n" ) ;
    return 1 ;
  }

  // XX
  FREAD32( &Grid -> nstpx , sizeof( int ) , 1 , fr ) ;
  double *XX = (double*)malloc( Grid -> nstpx * sizeof( double ) ) ;
  FREAD64( XX , sizeof( double ) , Grid -> nstpx , fr ) ;
  uint32_t cksumXX[2] = { 0 , 0 } , cksumXX_r[2] ;
  DML_checksum_accum_crc32c( &cksumXX[0] , &cksumXX[1] , 
			     0 , (char*)Grid -> XX ,
			     Grid -> nstpx*sizeof(double) ) ;
  FREAD32( cksumXX_r , sizeof( uint32_t ) , 2 , fr ) ;
  if( cksumXX[0] != cksumXX_r[0] || cksumXX[1] != cksumXX_r[1] ) {
    fprintf( stderr , "Computed and read XX checksums do not match!" ) ; 
    fprintf( stderr , "XX_a %u != %u\n" , cksumXX[0] , cksumXX_r[0] ) ;
    fprintf( stderr , "XX_b %u != %u\n" , cksumXX[1] , cksumXX_r[1] ) ;
    return 1 ;
  }
  checkCudaErrors(hipMalloc(&Grid->XX, Grid -> nstpx * sizeof( double )));
  checkCudaErrors(hipMemcpy(
      Grid->XX, XX, Grid -> nstpx * sizeof( double ), hipMemcpyHostToDevice));
  free(XX);

  // allocate nfx
  checkCudaErrors(hipMalloc(&Grid->nfx, Grid -> nstpx * sizeof( int ) )) ;
  int* nfx = (int*)malloc( Grid -> nstpx * sizeof( int ) ) ;

  // YY
  FREAD32( &Grid -> nstpy , sizeof( int ) , 1 , fr ) ;
  double *YY = (double*)malloc( Grid -> nstpy * sizeof( double ) ) ;
  FREAD64( YY , sizeof( double ) , Grid -> nstpy , fr ) ;
  uint32_t cksumYY[2] = { 0 , 0 } , cksumYY_r[2] ;
  DML_checksum_accum_crc32c( &cksumYY[0] , &cksumYY[1] , 
			     0 , (char*)Grid -> YY ,
			     Grid -> nstpy*sizeof(double) ) ;
  FREAD32( cksumYY_r , sizeof( uint32_t ) , 2 , fr ) ;
  if( cksumYY[0] != cksumYY_r[0] || cksumYY[1] != cksumYY_r[1] ) {
    fprintf( stderr , "Computed and read YY checksums do not match!" ) ; 
    fprintf( stderr , "XX_a %u != %u\n" , cksumYY[0] , cksumYY_r[0] ) ;
    fprintf( stderr , "XX_b %u != %u\n" , cksumYY[1] , cksumYY_r[1] ) ;
    return 1 ;
  }
  checkCudaErrors(hipMalloc(&Grid->YY, Grid -> nstpy * sizeof( double )));
  checkCudaErrors(hipMemcpy(
      Grid->YY, YY, Grid -> nstpy * sizeof( double ), hipMemcpyHostToDevice));
  free(YY);

#ifdef VERBOSE
  fprintf( stdout , "Nstpx %d :: Nstpy %d\n" , Grid -> nstpx , Grid -> nstpy ) ;
#endif
  
  Grid -> xstp = ( Grid -> XX[1] - Grid -> XX[0] ) ;
  Grid -> ystp = ( Grid -> YY[1] - Grid -> YY[0] ) ;    

#ifdef VERBOSE
  fprintf( stdout , "xstp %e :: ystp %e\n" , Grid -> xstp , Grid -> ystp ) ;
#endif
  
  FREAD32( &Grid -> Nffa , sizeof( int ) , 1 , fr ) ;
  Grid -> nfx_max = 0;

  float**** Ffm = (float****)malloc( Grid -> Nffa*sizeof(float***)) ;
  float**** Ffp = (float****)malloc( Grid -> Nffa*sizeof(float***)) ;
  
  int i , j , k , NX , NY , nx ;
  size_t rank = 0 ;
  uint32_t cksumFfm_a = 0 , cksumFfm_b = 0 ;
  uint32_t cksumFfp_a = 0 , cksumFfp_b = 0 ;
  for( i = 0 ; i < Grid -> Nffa ; i++ ) {
    FREAD32( &NX , sizeof( int ) , 1 , fr ) ;

    Ffm[i] = (float***)malloc( Grid -> nstpx*sizeof(float**)) ;
    Ffp[i] = (float***)malloc( Grid -> nstpx*sizeof(float**)) ; 

    if( NX != Grid -> nstpx ) {
      fprintf( stderr , "|IO| file is funny nstpx\n" ) ;
      return 1 ;
    }
    
    for( j = 0 ; j < NX ; j++ ) {
      FREAD32( &NY , sizeof( uint32_t ) , 1 , fr ) ;
      
      if( NY != Grid -> nstpy ) {
	fprintf( stderr , "|IO| file is funny nstpy\n" ) ;
	return 1 ;
      }

      Ffm[i][j] = (float**)malloc( Grid-> nstpy*sizeof(float*)) ;
      Ffp[i][j] = (float**)malloc( Grid-> nstpy*sizeof(float*)) ;
      
      for( k = 0 ; k < NY ; k++ ) {
	FREAD32( &nx , sizeof( int ) , 1 , fr ) ;

	Ffm[i][j][k] = (float*)malloc(nx*sizeof(float)) ;
	Ffp[i][j][k] = (float*)malloc(nx*sizeof(float)) ;

	// set the length of nfx
        if (k > 0 && nx != nfx[j]) {
          fprintf( stderr, "nx mismatch\n" );
          return 1 ;
        }
	nfx[j] = nx ;
        if (nx > Grid -> nfx_max) {
          Grid -> nfx_max = nx;
        }
	
	FREAD32( Ffm[i][j][k] , sizeof( float ) , nx , fr ) ;
	DML_checksum_accum_crc32c( &cksumFfm_a , &cksumFfm_b , 
				   rank , (char*) Ffm[i][j][k] ,
				   nx*sizeof(float) ) ;
	
	//
	FREAD32( Ffp[i][j][k] , sizeof( float ) , nx , fr ) ;
	DML_checksum_accum_crc32c( &cksumFfp_a , &cksumFfp_b , 
				   rank , (char*) Ffp[i][j][k] ,
				   nx*sizeof(float) ) ;
	//
	rank ++ ;
      }
    }
  }

  uint32_t cksum[4] ;
  FREAD32( cksum , sizeof( uint32_t ) , 4 , fr ) ;

  fclose( fr ) ;
  
  if( cksumFfm_a != cksum[0] || cksumFfm_b != cksum[1] ||
      cksumFfp_a != cksum[2] || cksumFfp_b != cksum[3] ) {
    fprintf( stderr , "Computed and read ff checksums do not match!" ) ; 
    fprintf( stderr , "Ffm_a %u != %u\n" , cksum[0] , cksumFfm_a ) ;
    fprintf( stderr , "Ffm_b %u != %u\n" , cksum[1] , cksumFfm_a ) ;
    fprintf( stderr , "Ffp_a %u != %u\n" , cksum[2] , cksumFfm_a ) ;
    fprintf( stderr , "Ffp_b %u != %u\n" , cksum[3] , cksumFfm_a ) ;
    return 1 ;
  }


  // build flattened rectangular array
  size_t sizeof_Ff = (
      Grid -> Nffa * Grid -> nstpx * Grid -> nstpy *
      Grid -> nfx_max * sizeof(float) );
  float* Ffp_rect = (float*)malloc( sizeof_Ff );
  float* Ffm_rect = (float*)malloc( sizeof_Ff );
  for( i = 0; i < Grid -> Nffa ; i++ ) {
    for( j = 0; j < Grid -> nstpx ; j++ ) {
      for( k = 0; k < Grid -> nstpy ; k++ ) {
        size_t ind = ((i * Grid -> nstpx + j) * Grid -> nstpy + k) * Grid -> nfx_max ;
        memcpy( &Ffp_rect[ind], Ffp[i][j][k], Grid -> nfx_max * sizeof(float) ) ;
        memcpy( &Ffm_rect[ind], Ffm[i][j][k], Grid -> nfx_max * sizeof(float) ) ;
        free( Ffp[i][j][k] ) ;
        free( Ffm[i][j][k] ) ;
      }
      free( Ffp[i][j] ) ;
      free( Ffm[i][j] ) ;
    }
    free( Ffp[i] ) ;
    free( Ffm[i] ) ;
  }
  free( Ffp );
  free( Ffm );

  checkCudaErrors(hipMalloc( &Grid -> Ffp, sizeof_Ff )) ;
  checkCudaErrors(hipMalloc( &Grid -> Ffm, sizeof_Ff )) ;
  checkCudaErrors(hipMemcpy(
      Grid -> Ffp, Ffp_rect, sizeof_Ff, hipMemcpyHostToDevice )) ;
  checkCudaErrors(hipMemcpy(
      Grid -> Ffm, Ffm_rect, sizeof_Ff, hipMemcpyHostToDevice )) ;
  free( Ffp_rect );
  free( Ffm_rect );

  checkCudaErrors(hipMemcpy(
      Grid -> nfx, nfx, Grid -> Nffa * sizeof(int), hipMemcpyHostToDevice )) ;
  free( nfx );

  return 0 ;
}

int
read_TAYLORX( struct Grid_coeffs *Grid )
{
  char filestr[ strlen( LIB_PATH ) + strlen( FILE_TAYLORX ) + 1 ] ;
  sprintf( filestr , "%s%s" , LIB_PATH , FILE_TAYLORX ) ;

  FILE *fr = fopen( filestr , "rb" ) ;
  if( fr == NULL ) {
    fprintf( stderr , "|IO| cannot open %s\n" , filestr ) ;
    return 1 ;
  }
  int moniker ;
  FREAD32( &moniker , sizeof( int ) , 1 , fr ) ;
  if( moniker != 816968 ) {
    fprintf( stderr , "|IO| misread magic number\n" ) ;
    return 1 ;
  }
  FREAD32( &Grid -> NtayY , sizeof( int ) , 1 , fr ) ;
  if( Grid -> NtayY != TX_LEN ) {
    fprintf( stderr , "|IO| taylorx.bin misread %d vs. %d\n" ,
	     Grid -> NtayY , TX_LEN ) ;
    return 1 ;
  }
  double *TX = (double*)malloc( TX_LEN * NYTAY * sizeof( double ) ) ;
  uint32_t cksuma = 0 , cksumb = 0 ;
  size_t i ;
  for( i = 0 ; i < (size_t)Grid -> NtayY ; i++ ) {
    FREAD32( &Grid -> NY_tay , sizeof( int ) , 1 , fr ) ;
    if( Grid -> NY_tay != NYTAY ) {
      fprintf( stderr , "|IO| taylorx.bin weird NY_TAY %d vs. %d\n" ,
	       Grid -> NY_tay, NYTAY ) ;
      return 1 ;
    }
    // TX[i] = malloc( Grid -> NY_tay * sizeof( double ) ) ;
    FREAD64( &TX[i * NYTAY] , sizeof( double ) , Grid -> NY_tay , fr ) ;
    DML_checksum_accum_crc32c( &cksuma , &cksumb , 
			       i , &TX[i * NYTAY] ,
			       Grid->NY_tay * sizeof(double) ) ;
  }
  checkCudaErrors(hipMalloc( &Grid -> TX, TX_LEN * NYTAY * sizeof( double ) )) ;
  checkCudaErrors(hipMemcpy(
      Grid -> TX, TX, TX_LEN * NYTAY * sizeof( double ), hipMemcpyHostToDevice )) ;
  free( TX );

  uint32_t cksum[2] ;
  FREAD32( cksum , sizeof( uint32_t ) , 2 , fr ) ;

  fclose( fr ) ;
  
  if( cksum[0] != cksuma || cksum[1] != cksumb ) {
    fprintf( stderr , "Read & Computed taylorx checksums do not match!" ) ; 
    fprintf( stderr , "Ffm_a %u != %u\n" , cksum[0] , cksuma ) ;
    fprintf( stderr , "Ffm_b %u != %u\n" , cksum[1] , cksumb ) ;
    return 1 ;
  }

  return 0 ;
}

// read the file taylory
int
read_TAYLORY( struct Grid_coeffs *Grid )
{
  char filestr[ strlen( LIB_PATH ) + strlen( FILE_TAYLORY ) + 1 ] ;
  sprintf( filestr , "%s%s" , LIB_PATH , FILE_TAYLORY ) ;

  FILE *fr = fopen( filestr , "rb" ) ;
  if( fr == NULL ) {
    fprintf( stderr , "|IO| cannot open %s\n" , filestr ) ;
    return 1 ;
  }

  int moniker ;
  FREAD32( &moniker , sizeof( int ) , 1 , fr ) ;
  if( moniker != 816968 ) {
    fprintf( stderr , "|IO| misread magic number\n" ) ;
    return 1 ;
  }
  FREAD32( &Grid -> NtayX , sizeof( int ) , 1 , fr ) ;
  if( Grid -> NtayX != 14 ) {
    fprintf( stderr , "|IO| taylory.bin misread %d vs. %d\n" ,
	     Grid -> NtayX , 14 ) ;
    return 1 ;
  }
  double *TY = (double*)malloc( TY_LEN * NXTAY * sizeof( double ) ) ;
  size_t i ;
  uint32_t cksuma = 0 , cksumb = 0 ;
  for( i = 0 ; i < (size_t)Grid -> NtayX ; i++ ) {
    FREAD32( &Grid -> NX_tay , sizeof( int ) , 1 , fr ) ;
    if( Grid -> NX_tay != NXTAY ) {
      fprintf( stderr , "|IO| taylory.bin weird NX_TAY %d vs. %d\n" ,
	       Grid -> NX_tay, NXTAY ) ;
      return 1 ;
    }
    // Grid -> TY[i] = malloc( Grid -> NX_tay * sizeof( double ) ) ;
    FREAD64( &Grid -> TY[i * NXTAY] , sizeof( double ) , Grid -> NX_tay , fr ) ;
    DML_checksum_accum_crc32c( &cksuma , &cksumb , 
			       i , &Grid -> TY[i * NXTAY] ,
			       Grid->NX_tay * sizeof(double) ) ;
  }
  checkCudaErrors(hipMalloc( &Grid -> TY, TY_LEN * NXTAY * sizeof( double ) )) ;
  checkCudaErrors(hipMemcpy(
      Grid -> TY, TY, TY_LEN * NXTAY * sizeof( double ), hipMemcpyHostToDevice )) ;
  free( TY );

  uint32_t cksum[2] ;
  FREAD32( cksum , sizeof( uint32_t ) , 2 , fr ) ;

  fclose( fr ) ;
  
  if( cksum[0] != cksuma || cksum[1] != cksumb ) {
    fprintf( stderr , "Read & Computed taylory checksums do not match!" ) ; 
    fprintf( stderr , "Ffm_a %u != %u\n" , cksum[0] , cksuma ) ;
    fprintf( stderr , "Ffm_b %u != %u\n" , cksum[1] , cksumb ) ;
    return 1 ;
  }
  
  return 0 ;
}

// write the form factor
void
write_ff( const struct Grid_coeffs *Grid )
{
  // write it out in one big file
  const int moniker = 816968 ;
  FILE *fw = fopen( "./PRECOMP/FFxy_single_cksum2.bin" , "wb" ) ;
  fwrite( &moniker , sizeof( int ) , 1 , fw ) ;
  
  fwrite( &Grid -> nstpx , sizeof( int ) , 1 , fw ) ;
  double *XX = (double*)malloc( Grid -> nstpx * sizeof( double ) ) ;
  checkCudaErrors(hipMemcpy(
      XX, Grid -> XX, Grid -> nstpx * sizeof( double ), hipMemcpyDeviceToHost )) ;
  fwrite( XX , sizeof( double ) , Grid -> nstpx , fw ) ;

  uint32_t cksumXX[2] = { 0 , 0 } ;
  DML_checksum_accum_crc32c( &cksumXX[0] , &cksumXX[1] , 
			     0 , (char*)XX ,
			     Grid -> nstpx*sizeof(double) ) ;
  fwrite( cksumXX , sizeof( uint32_t ) , 2 , fw ) ;
  free( XX ) ;
  
  fwrite( &Grid -> nstpy , sizeof( int ) , 1 , fw ) ;
  double *YY = (double*)malloc( Grid -> nstpy * sizeof( double ) ) ;
  checkCudaErrors(hipMemcpy(
      YY, Grid -> YY, Grid -> nstpy * sizeof( double ), hipMemcpyDeviceToHost )) ;
  fwrite( YY , sizeof( double ) , Grid -> nstpy , fw ) ;

  uint32_t cksumYY[2] = { 0 , 0 } ;
  DML_checksum_accum_crc32c( &cksumYY[0] , &cksumYY[1] ,
			     0 , (char*)YY ,
			     Grid -> nstpy*sizeof(double) ) ;
  fwrite( cksumYY , sizeof( uint32_t ) , 2 , fw ) ;
  free( YY ) ;
  
  int Nffa = Grid -> Nffa ;
  int i , j , k ;

  uint32_t cksumFfm_a = 0 , cksumFfm_b = 0 ;
  uint32_t cksumFfp_a = 0 , cksumFfp_b = 0 ;
  size_t rank = 0 ;

  size_t sizeof_Ff = (
      Grid -> Nffa * Grid -> nstpx * Grid -> nstpy *
      Grid -> nfx_max * sizeof(float) );
  float *Ffp_rect = (float*)malloc( sizeof_Ff );
  float *Ffm_rect = (float*)malloc( sizeof_Ff );
  int *nfx = (int*)malloc( Grid -> nstpx * sizeof( int ) ) ;
  checkCudaErrors(hipMemcpy(
      Ffp_rect, Grid -> Ffp, sizeof_Ff, hipMemcpyDeviceToHost )) ;
  checkCudaErrors(hipMemcpy(
      Ffm_rect, Grid -> Ffm, sizeof_Ff, hipMemcpyDeviceToHost )) ;
  checkCudaErrors(hipMemcpy(
      nfx, Grid -> nfx, Grid -> nstpx * sizeof( int ), hipMemcpyDeviceToHost )) ;
  fwrite( &Nffa , sizeof( int ) , 1 , fw ) ;
  for( i = 0 ; i < Nffa ; i++ ) {
    fwrite( &Grid -> nstpx , sizeof( int ) , 1 , fw ) ;
    for( j = 0 ; j < Grid -> nstpx ; j++ ) {
      int nx = nfx[j] ;
      fwrite( &Grid -> nstpy , sizeof( int ) , 1 , fw ) ;
      for( k = 0 ; k < Grid -> nstpy ; k++ ) {
	fwrite( &nx , sizeof( int ) , 1 , fw ) ;
        float* buf;
        buf = &Ffm_rect[((i*Grid->nstpx + j)*Grid->nstpy + k) * Grid -> nfx_max];
	fwrite( buf , sizeof( float ) , nx , fw ) ;
	DML_checksum_accum_crc32c( &cksumFfm_a , &cksumFfm_b , 
				   rank , (char*)buf , nx*sizeof(float) ) ;
        buf = &Ffp_rect[((i*Grid->nstpx + j)*Grid->nstpy + k) * Grid -> nfx_max];
	fwrite( buf , sizeof( float ) , nx , fw ) ;
	DML_checksum_accum_crc32c( &cksumFfp_a , &cksumFfp_b , 
				   rank , (char*)buf , nx*sizeof(float) ) ;
	rank++ ;
      }
    }
  }
  fwrite( &cksumFfm_a , sizeof( uint32_t ) , 1 , fw ) ;
  fwrite( &cksumFfm_b , sizeof( uint32_t ) , 1 , fw ) ;
  fwrite( &cksumFfp_a , sizeof( uint32_t ) , 1 , fw ) ;
  fwrite( &cksumFfp_b , sizeof( uint32_t ) , 1 , fw ) ;

  free( Ffp_rect );
  free( Ffm_rect );
  free( nfx );

  fclose( fw ) ;
  return ;
}

void
write_TAYLORX( const struct Grid_coeffs *Grid )
{
  FILE *fw = fopen( "./PRECOMP/taylorx_cksum.bin", "wb" ) ;
  if( fw == NULL ) {
    fprintf( stderr , "|IO| cannot write to file TAYLORX\n" ) ;
    return ;
  }
  const int moniker = 816968 ;
  fwrite( &moniker , sizeof( int ) , 1 , fw ) ;
  fwrite( &Grid -> NtayY , sizeof( int ) , 1 , fw ) ;
  uint32_t cksuma = 0 , cksumb = 0 ;
  size_t i ;
  double *TX  = (double*)malloc(Grid -> NtayY * Grid -> NY_tay * sizeof( double ) ) ;
  checkCudaErrors(hipMemcpy(
      TX, Grid -> TX, Grid -> NtayY * Grid -> NY_tay * sizeof( double ), hipMemcpyDeviceToHost )) ;
  for( i = 0 ; i < (size_t)Grid -> NtayY ; i++ ) {
    fwrite( &Grid -> NY_tay , sizeof( int ) , 1 , fw ) ;
    fwrite( &TX[i * Grid -> NY_tay] , sizeof( double ) , Grid -> NY_tay , fw  ) ;
    DML_checksum_accum_crc32c( &cksuma , &cksumb , 
			       i , &Grid -> TX[i * Grid -> NY_tay] ,
			       Grid->NY_tay * sizeof(double) ) ;
  }
  free( TX ) ;
  fwrite( &cksuma , sizeof( uint32_t ) , 1 , fw ) ;
  fwrite( &cksumb , sizeof( uint32_t ) , 1 , fw ) ;
  fclose( fw ) ;
  return ;
}

void
write_TAYLORY( const struct Grid_coeffs *Grid )
{  
  FILE *fw = fopen( "./PRECOMP/taylory_cksum.bin", "wb" ) ;
  if( fw == NULL ) {
    fprintf( stderr , "|IO| cannot write to file TAYLORY\n" ) ;
    return ;
  }
  const int moniker = 816968 ;
  fwrite( &moniker , sizeof( int ) , 1 , fw ) ;
  fwrite( &Grid -> NtayX , sizeof( int ) , 1 , fw ) ;
  
  uint32_t cksuma = 0 , cksumb = 0 ;
  size_t i ;
  double *TY  = (double*)malloc(Grid -> NtayX * Grid -> NX_tay * sizeof( double ) ) ;
  checkCudaErrors(hipMemcpy(
      TY, Grid -> TY, Grid -> NtayX * Grid -> NX_tay * sizeof( double ), hipMemcpyDeviceToHost )) ;
  for( i = 0 ; i < (size_t)Grid -> NtayX ; i++ ) {
    fwrite( &Grid -> NX_tay , sizeof( int ) , 1 , fw ) ;
    fwrite( &TY[i * Grid -> NX_tay] , sizeof( double ) , Grid -> NX_tay , fw  ) ;
    DML_checksum_accum_crc32c( &cksuma , &cksumb , 
			       i , &TY[i * Grid -> NX_tay] ,
			       Grid->NX_tay * sizeof(double) ) ;
  }
  free( TY ) ;
  
  fwrite( &cksuma , sizeof( uint32_t ) , 1 , fw ) ;
  fwrite( &cksumb , sizeof( uint32_t ) , 1 , fw ) ;
  
  fclose( fw ) ;

  return ;
}

#ifdef FILE_FF
  #undef FILE_FF
#endif

#ifdef FILE_TAYLORX
  #undef FILE_TAYLORX
#endif

#ifdef FILE_TAYLORY
  #undef FILE_TAYLORY
#endif
