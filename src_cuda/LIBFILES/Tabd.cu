#include "hip/hip_runtime.h"
/**
   @file Tabd.c
   @brief contains the functions Tabd_xeq0 and Tabd_yeq0

   @warning these differ by a factor of 2 from the old code as they contain the implicit SYMG normalisation
 */
#include "KQED.h"

#include "chnr_dV.h"   // lerp
#include "getff-new.h" // SCALPROD, bsrch

// delta function
__device__
static const double dlta[4][4] = { { 1 , 0 , 0 , 0 } ,
				   { 0 , 1 , 0 , 0 } ,
				   { 0 , 0 , 1 , 0 } ,
				   { 0 , 0 , 0 , 1 } } ;

// Taylor expanded versions for x == 0
__device__ KQED_PRIVATE
int
Tabd_xeq0( const double yv[4] ,
	   const struct Grid_coeffs Grid ,
	   double tI[4][4][4] ,
	   double tII[4][4][4] ,
	   double tIII[4][4][4] )
{ 
  const double ysq = SCALPROD(yv,yv);
  const double y = sqrt( fabs( ysq ) );
  
  const int iy_tay = find_ind( getTX(&Grid,YY) , y , 0 , Grid.NY_tay ) ;

  if( iy_tay == Grid.NY_tay-1 ) {
    // need to return here otherwise we will segfault
    return 1 ;
  }
  // can set this after we checked the above error
  const int iy2_tay = iy_tay+1 ;
  
  const double ay = (getTX(&Grid,YY)[iy2_tay]-y)/( getTX(&Grid,YY)[iy2_tay]-getTX(&Grid,YY)[iy_tay]);
  
  const double fx       = 0.5*lerp( ay , getTX(&Grid,G0dx)[iy_tay] , getTX(&Grid,G0dx)[iy2_tay] ) ; 
  const double fy       = 0.5*lerp( ay , getTX(&Grid,G0dy)[iy_tay] , getTX(&Grid,G0dy)[iy2_tay] ) ;
  const double yhat[4]  = { yv[0]/y , yv[1]/y , yv[2]/y , yv[3]/y } ;
  
  const double dg0dy[4] = { yhat[0]*fy/2 , yhat[1]*fy/2 , yhat[2]*fy/2 , yhat[3]*fy/2 } ; 
  const double dg0dx[4] = { yhat[0]*fx/2 , yhat[1]*fx/2 , yhat[2]*fx/2 , yhat[3]*fx/2 } ;
  
  const double ell2     = lerp( ay , getTX(&Grid,Gl2)[iy_tay] , getTX(&Grid,Gl2)[iy2_tay] ) ; 
  const double dell2adx = lerp( ay , getTX(&Grid,Gl21)[iy_tay] , getTX(&Grid,Gl21)[iy2_tay] ) ;
  const double dell2ady = lerp( ay , getTX(&Grid,Gl2dy)[iy_tay] , getTX(&Grid,Gl2dy)[iy2_tay] ) ;
  const double ell3a    = lerp( ay , getTX(&Grid,Gl3)[iy_tay] , getTX(&Grid,Gl3)[iy2_tay] ) ;
  
  const double dg2dx    = lerp( ay , getTX(&Grid,G21)[iy_tay] , getTX(&Grid,G21)[iy2_tay] ) ;
  const double ddg2dxdy = lerp( ay , getTX(&Grid,G21dy)[iy_tay] , getTX(&Grid,G21dy)[iy2_tay] ) ;
  const double dg1dx    = (lerp( ay , getTX(&Grid,G31A)[iy_tay] , getTX(&Grid,G31A)[iy2_tay] )
			   -(2./3)*( lerp( ay , getTX(&Grid,G31B)[iy_tay] , getTX(&Grid,G31B)[iy2_tay] ) )
			   -y* ( lerp( ay , getTX(&Grid,G22A)[iy_tay] , getTX(&Grid,G22A)[iy2_tay] ) ) ) ;
  const double dg1dy    = (lerp( ay , getTX(&Grid,G3Ady)[iy_tay] , getTX(&Grid,G3Ady)[iy2_tay] )
			   -lerp( ay , getTX(&Grid,G3Bdy)[iy_tay] , getTX(&Grid,G3Bdy)[iy2_tay] ) ) ;
  const double phi1     = ( lerp( ay , getTX(&Grid,G22A)[iy_tay] , getTX(&Grid,G22A)[iy2_tay] )
			    - lerp( ay , getTX(&Grid,G22B)[iy_tay] , getTX(&Grid,G22B)[iy2_tay] )/5. ) ;
  const double phi2     = ( lerp( ay , getTX(&Grid,G22B)[iy_tay] , getTX(&Grid,G22B)[iy2_tay] ) )*1.2 ;
  
  // various precomputations to make the loops faster
  const double ysq_4 = ysq/4. ;
  const double E = (ell3a + ell2) , D2 = (dell2adx + dell2ady) ,
    D1 = (dg1dy+dg1dx) , D12 = (dg1dx+dg2dx) , D3 = (y*ddg2dxdy-dg2dx) ;
  const double yvE[4]       = { yv[0]*E , yv[1]*E , yv[2]*E , yv[3]*E } ;
  const double yvell3a[4]   = { yv[0]*ell3a , yv[1]*ell3a , yv[2]*ell3a , yv[3]*ell3a } ;
  const double yhatD1[4]    = { yhat[0]*D1 , yhat[1]*D1 , yhat[2]*D1 , yhat[3]*D1 } ;
  const double yhatD2[4]    = { yhat[0]*D2 , yhat[1]*D2 , yhat[2]*D2 , yhat[3]*D2 } ;
  const double yhatD3[4]    = { yhat[0]*D3 , yhat[1]*D3 , yhat[2]*D3 , yhat[3]*D3 } ;
  const double yhatdg2dx[4] = { yhat[0]*dg2dx , yhat[1]*dg2dx , yhat[2]*dg2dx , yhat[3]*dg2dx } ;
  
  // point out the structures
  double *PtIII = (double*)tIII ;
  double *PtII  = (double*)tII ;
  double *PtI   = (double*)tI ;
  
  int alf , dta , bet ;
  for(alf=0;alf<4;alf++) {
    const double yalf2dx = yhat[alf]*(dell2adx) ;
    const double tIIsum  = dg0dx[alf]-0.5*yvell3a[alf]-ysq_4*yalf2dx;
    const double yD12    = yhat[alf]*D12 ;
    for(bet=0;bet<4;bet++) {
      const double dgSUM   = (dg0dx[bet]+dg0dy[bet]) ; 
      const double tIsum   = 2*(dlta[alf][bet]*phi1+yhat[alf]*yhat[bet]*phi2) ;
      const double tIIIsum = dgSUM-0.5*yvE[bet]-ysq_4*yhatD2[bet] ;
      for(dta=0;dta<4;dta++) {
	// (d/dxbeta+d/dybeta) T_{alpha delta}
	*PtIII = (yv[alf]*yv[dta]*yhatD2[bet]
		  +dlta[alf][bet]*yvE[dta]
		  +dlta[bet][dta]*yvE[alf]
		  +dlta[alf][dta]*tIIIsum) ; PtIII++ ;
	// d/dxalpha T_{beta delta}
	*PtII = (yv[bet]*yv[dta]*yalf2dx+dlta[bet][dta]*(tIIsum) 
		 +(dlta[bet][alf]*yvell3a[dta]+dlta[alf][dta]*yvell3a[bet])) ; PtII++ ;
	// first the d/dxalpha d/dybeta V_delta terms:  
        *PtI = (dlta[bet][dta]*yD12+yhatdg2dx[dta]*dlta[alf][bet]
		+yhatD3[bet]*yhat[alf]*yhat[dta]
		+yhatD1[bet]*dlta[alf][dta]+yv[dta]*tIsum) ; PtI++ ;
      }
    }
  }
    
  return 0 ;
}

// Taylor expansions around y = 0
__device__ KQED_PRIVATE
int
Tabd_yeq0( const double xv[4] ,
	   const struct Grid_coeffs Grid ,
	   double tI[4][4][4] ,
	   double tII[4][4][4] ,
	   double tIII[4][4][4] )
{  
  const double xsq = SCALPROD(xv,xv);
  const double x = sqrt( fabs(xsq) ); 

  const int ix_tay = find_ind( Grid.XX , x , 0 , Grid.nstpx ) ;
  
  if( x > Grid.XX[ Grid.nstpx - 1 ] ) {
    // need to return here otherwise we will segfault
    return 1 ;
  }
  // we can set this now as we aren't at the upper edge
  const int ix2_tay = ix_tay+1 ;
  
  const double ax = ( Grid.XX[ix2_tay]-x)/( Grid.XX[ix2_tay]-Grid.XX[ix_tay]);
  const double xa = Grid.XX[ix_tay];
  const double xb = Grid.XX[ix2_tay];
  const double fx = 0.5*lerp( ax , getTY(&Grid,alpha0dx_0p)[ ix_tay ], getTY(&Grid,alpha0dx_0p)[ix2_tay] ) ;
  const double fy = ( lerp( ax , getTY(&Grid,alpha0_1p)[ix_tay] , getTY(&Grid,alpha0_1p)[ix2_tay] ) );

  // set by hand the array variables
  const double xhat[4] = { xv[0]/x , xv[1]/x , xv[2]/x , xv[3]/x } ;
  const double dg0dy[4] = { xhat[0]*fy/2 , xhat[1]*fy/2 , xhat[2]*fy/2 , xhat[3]*fy/2 } ;
  const double dg0dx[4] = { xhat[0]*fx/2 , xhat[1]*fx/2 , xhat[2]*fx/2 , xhat[3]*fx/2 } ;

  const double xa2 = xa*xa ;
  const double xa3 = xa2*xa ;
  const double xa4 = xa3*xa ;
  const double xb2 = xb*xb ;
  const double xb3 = xb2*xb ;
  const double xb4 = xb3*xb ;

  const double ell1       = 4*( lerp( ax , getTY(&Grid,alpha1_0p)[ix_tay]/(xa4) ,
				      getTY(&Grid,alpha1_0p)[ix2_tay]/(xb4) ) )/3. ;
  const double ell3       = lerp( ax , getTY(&Grid,beta4_1p)[ix_tay]/(xa2) ,
				  getTY(&Grid,beta4_1p)[ix2_tay]/(xb2) ) ;
  
  const double dell1dx    = 4*lerp( ax , (getTY(&Grid,alpha1dx_0p)[ix_tay]
					  -4.*getTY(&Grid,alpha1_0p)[ix_tay]/xa)/xa4 ,
				    (getTY(&Grid,alpha1dx_0p)[ix2_tay]
				     -4.*getTY(&Grid,alpha1_0p)[ix2_tay]/xb)/xb4 )/3. ;
  
  const double dell1dycb  = 2*lerp( ax , (4.*getTY(&Grid,alpha1_1p)[ix_tay]/(3*xa)
					  - getTY(&Grid,beta4_1p)[ix_tay])/xa3 ,
				    (4.*getTY(&Grid,alpha1_1p)[ix2_tay]/(3*xb)
				     - getTY(&Grid,beta4_1p)[ix2_tay])/xb3 ) ;
  
  const double dg1dx      = lerp( ax , getTY(&Grid,alpha3dx_0p)[ix_tay] , getTY(&Grid,alpha3dx_0p)[ix2_tay] ) ;
  const double ddg1dxdx   = lerp( ax , getTY(&Grid,alpha3dxdx_0p)[ix_tay] , getTY(&Grid,alpha3dxdx_0p)[ix2_tay] ) ;
  const double dg2dx      = 2*lerp( ax , getTY(&Grid,beta2dx_1p)[ix_tay] , getTY(&Grid,beta2dx_1p)[ix2_tay] ) ;
  
  const double dg1dycb    = 2*lerp( ax , getTY(&Grid,alpha3_1p)[ix_tay]
				    - getTY(&Grid,beta2_1p)[ix_tay]/xa ,
				    getTY(&Grid,alpha3_1p)[ix2_tay]
				    - getTY(&Grid,beta2_1p)[ix2_tay]/xb ) ;
  
  const double ddg1dxdycb = 2*lerp( ax , getTY(&Grid,beta2_1p)[ix_tay]/(xa2)
				    - getTY(&Grid,beta2dx_1p)[ix_tay]/xa
				    + getTY(&Grid,alpha3dx_1p)[ix_tay] ,
				    getTY(&Grid,beta2_1p)[ix2_tay]/(xb2)
				    - getTY(&Grid,beta2dx_1p)[ix2_tay]/xb
				    + getTY(&Grid,alpha3dx_1p)[ix2_tay] ) ;

  // precomputations
  const double xsq_4 = xsq/4. , E = ell1+ell3 , D1 = dell1dx + dell1dycb
    , D2 = (ddg1dxdx+ddg1dxdycb) , D3 = (dg1dycb-dg2dx) , D4 = (dg1dx+dg1dycb) ;
  const double xvE[4]    = { xv[0]*E , xv[1]*E , xv[2]*E , xv[3]*E } ;
  const double xvell1[4] = { xv[0]*ell1 , xv[1]*ell1 , xv[2]*ell1 , xv[3]*ell1 } ;

  // point out the structures
  double *PtIII = (double*)tIII ;
  double *PtII  = (double*)tII ;
  double *PtI   = (double*)tI ;

  int alf , dta , bet ;
  for(alf=0;alf<4;alf++) {
    const double xhd1dx = xhat[alf]*(dell1dx) ;
    const double tIIsum = (dg0dx[alf]-0.5*xvell1[alf] ); 
    for(bet=0;bet<4;bet++) {
      const double Dg0SUM = (dg0dx[bet]+dg0dy[bet]-0.5*xvE[bet]) ;
      for(dta=0;dta<4;dta++) {
	const double xad = (xv[alf]*xv[dta]-xsq_4*dlta[alf][dta]);
	*PtIII = ((xad*xhat[bet]*D1 + dlta[alf][bet]*xvE[dta]+dlta[bet][dta]*xvE[alf]+Dg0SUM*dlta[alf][dta]) ); PtIII++ ;
	*PtII  = ((xv[bet]*xv[dta]-xsq_4*dlta[bet][dta])*xhd1dx+dlta[bet][alf]*xvell1[dta]+dlta[alf][dta]*xvell1[bet]+dlta[bet][dta]*(tIIsum)); PtII++ ;
	*PtI   = ((dlta[alf][dta]*xhat[bet]+dlta[bet][dta]*xhat[alf]+dlta[alf][bet]*xhat[dta]-xhat[alf]*xhat[bet]*xhat[dta])*D4+xhat[alf]*( xhat[bet]*xv[dta]*D2 - dlta[bet][dta]*D3 ));PtI++ ;	
      }
    }
  }
  
  return 0 ;
}
