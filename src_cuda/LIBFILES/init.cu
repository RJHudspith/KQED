/**
   @file init.c
   @brief a place where things are initialised
 */
#include "KQED.h"

#include "io.h"        // read_ff()
#include "pi_pert.h"   // init_g8()

// set the Grid struct and all the grid information e.g. XX,YY.
// this includes uploading grid info to GPU, though Grid is still a host struct
// wrapping the device ptrs.
__host__
static int
set_grid( struct Grid_coeffs *Grid )
{  
  if( read_ff( Grid ) ) {
    return 1 ;
  }
  if( read_TAYLORX( Grid ) ) {
    return 1 ;
  }
  if( read_TAYLORY( Grid ) ) {
    return 1 ;
  }

#ifdef VERBOSE
  fprintf( stdout , "Successfully Initialized QED kernel FFs...\n");
#endif
  
  return 0 ;
}

// free the struct t
void
free_QED_temps( struct QED_kernel_temps *t )
{
  // free these grid parameters
  if( t->Grid.XX != NULL ) {
    checkCudaErrors(hipFree( t->Grid.XX )) ;
  }
  if( t->Grid.nfx != NULL ) {
    checkCudaErrors(hipFree( t->Grid.nfx )) ;
  }
  if( t[0].Grid.YY != NULL ) {
    checkCudaErrors(hipFree( t->Grid.YY )) ;
  }

  // Free Ffm
  if( t->Grid.Ffm != NULL ) {
    checkCudaErrors(hipFree( t->Grid.Ffm )) ;
  }

  // Free Ffp
  if( t->Grid.Ffp != NULL ) {
    checkCudaErrors(hipFree( t->Grid.Ffp )) ;
  }
  
  // free the Taylor coefficients
  if( t->Grid.TX != NULL ) {
    checkCudaErrors(hipFree( t->Grid.TX )) ;
  }
  
  if( t->Grid.TY != NULL ) {
    checkCudaErrors(hipFree( t->Grid.TY )) ;
  }

  // free G8
  if( t->G8 != NULL ) {
    checkCudaErrors(hipFree( t->G8 )) ;
  }
  
  return ;
}

// returns 1 if it messes up, returns 0 otherwise
__host__
int
initialise( struct QED_kernel_temps *t )
{
  // set the grid struct and initialise the ff arrays
  if( set_grid( &( t -> Grid ) ) ) {
    return 1 ;
  }

  checkCudaErrors(hipMalloc((void**)&t->G8, 65536 * sizeof( int ) )) ;
  init_g8( t );

#ifdef VERBOSE
  fprintf( stdout , "|INIT| Set g-factors\n") ;
#endif

  return 0 ;
}
