#include "hip/hip_runtime.h"
/**
   @file all_kernels.c
   @brief compute all kernels L^{0/1/2/3} for (x,y) and (y,x) arguements

   Also contains the average function for all kernels
 */
#include "KQED.h"

#include "kernels.h"

// from linear idx in 0->384 return a value where mu and lambda indices are swapped
__device__
static inline size_t
i_to_mulam( const size_t idx )
{
  const size_t l[4] = { idx/64 , (idx/16)&3 , (idx/4)&3 , idx&3 } ;
  return l[1] + 4*(l[2]+4*(l[3]+4*l[0]) ) ;
}

// computes *K = 0.5*( *K + *W ) for all kernels in K and W
__device__
void
average_all_kernels( struct Kernels *K ,
		     const struct Kernels W )
{
  // point out elements of K
  double *KL0 = (double*)K->L0 , *KL1 = (double*)K->L1 ;
  double *KL2 = (double*)K->L2 , *KL3 = (double*)K->L3 ;
  
  // const point out elements of W
  const double *WL0 = (const double*)W.L0 ;
  const double *WL1 = (const double*)W.L1 ;
  const double *WL2 = (const double*)W.L2 ;
  const double *WL3 = (const double*)W.L3 ;

  // do the average
  size_t i ;
  for( i = 0 ; i < 384 ; i++ ) {
    *KL0 = 0.5*( *KL0 + *WL0 ) ; KL0++ ; WL0++ ;
    *KL1 = 0.5*( *KL1 + *WL1 ) ; KL1++ ; WL1++ ;
    *KL2 = 0.5*( *KL2 + *WL2 ) ; KL2++ ; WL2++ ;
    *KL3 = 0.5*( *KL3 + *WL3 ) ; KL3++ ; WL3++ ;
  }
  return ;
}

// computes *K = 0.5*( *K + *W ) for all kernels in K and W
__device__
void
average_all_QED_kernels( struct QED_Kernels *K ,
			 const struct QED_Kernels W )
{
  // point out elements of K
  double *KL0xy = (double*)K->L0.xy ;
  double *KL0yx = (double*)K->L0.yx ;
  double *KL1xy = (double*)K->L1.xy ;
  double *KL1yx = (double*)K->L1.yx ;
  double *KL2xy = (double*)K->L2.xy ;
  double *KL2yx = (double*)K->L2.yx ;
  double *KL3xy = (double*)K->L3.xy ;
  double *KL3yx = (double*)K->L3.yx ;
  
  // const point out elements of W
  const double *WL0xy = (const double*)W.L0.xy ;
  const double *WL0yx = (const double*)W.L0.yx ;
  const double *WL1xy = (const double*)W.L1.xy ;
  const double *WL1yx = (const double*)W.L1.yx ;
  const double *WL2xy = (const double*)W.L2.xy ;
  const double *WL2yx = (const double*)W.L2.yx ;
  const double *WL3xy = (const double*)W.L3.xy ;
  const double *WL3yx = (const double*)W.L3.yx ;

  // do the average
  size_t i ;
  for( i = 0 ; i < 384 ; i++ ) {
    *KL0xy = 0.5*( *KL0xy + *WL0xy ) ; KL0xy++ ; WL0xy++ ;
    *KL0yx = 0.5*( *KL0yx + *WL0yx ) ; KL0yx++ ; WL0yx++ ;
    *KL1xy = 0.5*( *KL1xy + *WL1xy ) ; KL1xy++ ; WL1xy++ ;
    *KL1yx = 0.5*( *KL1yx + *WL1yx ) ; KL1yx++ ; WL1yx++ ;
    *KL2xy = 0.5*( *KL2xy + *WL2xy ) ; KL2xy++ ; WL2xy++ ;
    *KL2yx = 0.5*( *KL2yx + *WL2yx ) ; KL2yx++ ; WL2yx++ ;
    *KL3xy = 0.5*( *KL3xy + *WL3xy ) ; KL3xy++ ; WL3xy++ ;
    *KL3yx = 0.5*( *KL3yx + *WL3yx ) ; KL3yx++ ; WL3yx++ ;
  }
  return ;
}

// computes L_{rhosig;mu,nu,lambda}(x,y) and L_{rhosig;mu,nu,lambda}(y,x)
__device__
void
compute_all_kernels( const double xv[4] ,
		     const double yv[4] ,
		     const struct QED_kernel_temps t ,
		     struct QED_Kernels *K )
{
  // we only need 6 kernels to compute everything
  // K[0].Lxy = L(x,y) K[0].Lyx = L(y,x)
  QED_kernel_L0( xv , yv , t , K -> L0.xy ) ;
  QED_kernel_L0( yv , xv , t , K -> L0.yx ) ;

  // and these temporaries: L(x,0) L(y,0) L(0,x) L(0,y)
  double Lx0[6][4][4][4] , L0x[6][4][4][4] ;
  double Ly0[6][4][4][4] , L0y[6][4][4][4] ;
  const double zero[4] = { 0. , 0. , 0. , 0. } ;

  QED_kernel_L0( xv , zero , t , Lx0 ) ;
  QED_kernel_L0( zero , xv , t , L0x ) ;

  QED_kernel_L0( yv , zero , t , Ly0 ) ;
  QED_kernel_L0( zero , yv , t , L0y ) ;

  // point out the data structures we want
  const double *L0xy   = (const double*)K->L0.xy ;
  const double *L0yx   = (const double*)K->L0.yx ;
  const double *pLx0   = (const double*)Lx0 ;
  const double *pLy0   = (const double*)Ly0 ;
  const double *pLx0ml = (const double*)Lx0 ;
  const double *pLy0ml = (const double*)Ly0 ;
  const double *pL0x   = (const double*)L0x ;
  const double *pL0y   = (const double*)L0y ;

  // write into these structures
  double *L1xy = (double*)K->L1.xy ;
  double *L2xy = (double*)K->L2.xy ;
  double *L3xy = (double*)K->L3.xy ;
  double *L1yx = (double*)K->L1.yx ;
  double *L2yx = (double*)K->L2.yx ;
  double *L3yx = (double*)K->L3.yx ;
  
  size_t i ;
  for( i = 0 ; i < 384 ; i++ ) {

    const size_t idx = i_to_mulam(i) ;
    const double Lx0mulam = *(pLx0ml + idx) ;
    const double Ly0mulam = *(pLy0ml + idx) ;
    const double f1 = ( Lx0mulam + Ly0mulam )/2. ;
    const double f2 = ( *pLx0 + *pL0y ) ;
    const double f4 = ( *pL0x + *pLy0 ) ;
    const double f3 = ( *pL0x - *pL0y ) ;

    // L^(1)(x,y)
    *L1xy = *L0xy + f1 ;
    // L^(2)(x,y)
    *L2xy = *L0xy - f2 ;
    // L^3(x,y)
    *L3xy = *L0xy + Lx0mulam + f3 ;
    // L^1(y,x)
    *L1yx = *L0yx + f1 ;
    // L^2(y,x)
    *L2yx = *L0yx - f4 ;
    // L^3(y,x)
    *L3yx = *L0yx + Ly0mulam - f3 ;
    
    // increment pointers
    L0xy++ ; L1xy++ ; L2xy++ ; L3xy++ ;
    L0yx++ ; L1yx++ ; L2yx++ ; L3yx++ ;
    pLx0 ++ ; pLy0++ ; pL0x++ ; pL0y++ ;
  }
  
  return ;
}

// copy for the XY
#define inlineXY(a)							\
  ( + eX[a]*(L0y[rhosig][mu][nu][lambda] -				\
	     Mx[mu][a]*(xv[0]*L0y[rhosig][0][nu][lambda] +		\
			xv[1]*L0y[rhosig][1][nu][lambda] +		\
			xv[2]*L0y[rhosig][2][nu][lambda] +		\
			xv[3]*L0y[rhosig][3][nu][lambda] ) )		\
    + eY[a]*(Lx0[rhosig][mu][nu][lambda] -				\
	     My[nu][a]*(yv[0]*Lx0[rhosig][mu][0][lambda] +		\
			yv[1]*Lx0[rhosig][mu][1][lambda] +		\
			yv[2]*Lx0[rhosig][mu][2][lambda] +		\
			yv[3]*Lx0[rhosig][mu][3][lambda] ) ) )

// copy for the XY
#define inlineXMY(a)							\
  ( - eX[a]*(L0y[rhosig][mu][nu][lambda] -				\
	     Mx[mu][a]*(xv[0]*L0y[rhosig][0][nu][lambda] +		\
			xv[1]*L0y[rhosig][1][nu][lambda] +		\
			xv[2]*L0y[rhosig][2][nu][lambda] +		\
			xv[3]*L0y[rhosig][3][nu][lambda] ) )		\
    + eY[a]*(Lx0[rhosig][mu][nu][lambda] -				\
	     My[nu][a]*(yv[0]*Lx0[rhosig][mu][0][lambda] +		\
			yv[1]*Lx0[rhosig][mu][1][lambda] +		\
			yv[2]*Lx0[rhosig][mu][2][lambda] +		\
			yv[3]*Lx0[rhosig][mu][3][lambda] ) ) )

// copy for the YX
#define inlineYX(a)				\
  ( + eY[a]*(L0x[rhosig][mu][nu][lambda] -				\
	   My[mu][a]*(yv[0]*L0x[rhosig][0][nu][lambda] +		\
		      yv[1]*L0x[rhosig][1][nu][lambda] +		\
		      yv[2]*L0x[rhosig][2][nu][lambda] +		\
		      yv[3]*L0x[rhosig][3][nu][lambda] ) )		\
    + eX[a]*(Ly0[rhosig][mu][nu][lambda] -				\
	     Mx[nu][a]*(xv[0]*Ly0[rhosig][mu][0][lambda] +		\
			xv[1]*Ly0[rhosig][mu][1][lambda] +		\
			xv[2]*Ly0[rhosig][mu][2][lambda] +		\
			xv[3]*Ly0[rhosig][mu][3][lambda] ) ) )

// copy for the YX
#define inlineYX2(a)							\
  ( + eY[a]*(L0x[rhosig][nu][mu][lambda] -				\
	   My[nu][a]*(yv[0]*L0x[rhosig][0][mu][lambda] +		\
		      yv[1]*L0x[rhosig][1][mu][lambda] +		\
		      yv[2]*L0x[rhosig][2][mu][lambda] +		\
		      yv[3]*L0x[rhosig][3][mu][lambda] ) )		\
    + eX[a]*(Ly0[rhosig][nu][mu][lambda] -				\
	     Mx[mu][a]*(xv[0]*Ly0[rhosig][nu][0][lambda] +		\
			xv[1]*Ly0[rhosig][nu][1][lambda] +		\
			xv[2]*Ly0[rhosig][nu][2][lambda] +		\
			xv[3]*Ly0[rhosig][nu][3][lambda] ) ) )

// computes L_{rhosig;mu,nu,lambda}(x,y) and L_{rhosig;mu,nu,lambda}(y,x)
__device__
void
compute_all_Mkernels( const double M[4] ,
		      const double xv[4] ,
		      const double yv[4] ,
		      const struct QED_kernel_temps t ,
		      struct QED_Kernels *K )
{
  // we only need 6 kernels to compute everything
  double Lxy[6][4][4][4] , Lyx[6][4][4][4] ;
  QED_kernel_L0( xv , yv , t , Lxy ) ;
  QED_kernel_L0( yv , xv , t , Lyx ) ;

  // and these temporaries: L(x,0) L(y,0) L(0,x) L(0,y)
  double Lx0[6][4][4][4] , L0x[6][4][4][4] ;
  double Ly0[6][4][4][4] , L0y[6][4][4][4] ;
  const double zero[4] = { 0. , 0. , 0. , 0. } ;

  QED_kernel_L0( xv , zero , t , Lx0 ) ;
  QED_kernel_L0( zero , xv , t , L0x ) ;

  QED_kernel_L0( yv , zero , t , Ly0 ) ;
  QED_kernel_L0( zero , yv , t , L0y ) ;

    // precompute x^2, y^2
  const double xsq = xv[0]*xv[0]+xv[1]*xv[1]+xv[2]*xv[2]+xv[3]*xv[3] ;
  const double ysq = yv[0]*yv[0]+yv[1]*yv[1]+yv[2]*yv[2]+yv[3]*yv[3] ;

  // precompute the gaussians
  const double eX[4] = { exp(-M[0]*xsq/2.) , exp(-M[1]*xsq/2.) ,
			 exp(-M[2]*xsq/2.) , exp(-M[3]*xsq/2.) } ;
  const double eY[4] = { exp(-M[0]*ysq/2.) , exp(-M[1]*ysq/2.) ,
			 exp(-M[2]*ysq/2.) , exp(-M[3]*ysq/2.) } ;

  // look-up table precomputations
  const double Mx[4][4] = { { M[0]*xv[0] , M[1]*xv[0] , M[2]*xv[0] , M[3]*xv[0] } ,
			    { M[0]*xv[1] , M[1]*xv[1] , M[2]*xv[1] , M[3]*xv[1] } ,
			    { M[0]*xv[2] , M[1]*xv[2] , M[2]*xv[2] , M[3]*xv[2] } ,
			    { M[0]*xv[3] , M[1]*xv[3] , M[2]*xv[3] , M[3]*xv[3] } } ;
  const double My[4][4] = { { M[0]*yv[0] , M[1]*yv[0] , M[2]*yv[0] , M[3]*yv[0] } ,
			    { M[0]*yv[1] , M[1]*yv[1] , M[2]*yv[1] , M[3]*yv[1] } ,
			    { M[0]*yv[2] , M[1]*yv[2] , M[2]*yv[2] , M[3]*yv[2] } ,
			    { M[0]*yv[3] , M[1]*yv[3] , M[2]*yv[3] , M[3]*yv[3] } } ;

  size_t rhosig , mu , nu , lambda ;
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {
      for( nu = 0 ; nu < 4 ; nu++ ) {
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {
	  // M0
	  K->L0.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(0) ;
	  K->L0.yx[rhosig][mu][nu][lambda] =
	    Lyx[rhosig][mu][nu][lambda] - inlineYX(0) ;
	  // M1
	  K->L1.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(1) ;
	  K->L1.yx[rhosig][mu][nu][lambda] =
	    Lyx[rhosig][mu][nu][lambda] - inlineYX(1) ;
	  // M2
	  K->L2.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(2) ;
	  K->L2.yx[rhosig][mu][nu][lambda] =
	    Lyx[rhosig][mu][nu][lambda] - inlineYX(2) ;
	  // M3
	  K->L3.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(3) ;
	  K->L3.yx[rhosig][mu][nu][lambda] =
	    Lyx[rhosig][mu][nu][lambda] - inlineYX(3) ;
	}
      }
    }
  }
  return ;
}

// computes L_{rhosig;mu,nu,lambda}(x,y) + L_{rhosig;nu,mu,lambda}(y,x) in Lxy
// and puts L_{rhosig;mu,nu,lambda}(x,-y) in Lyx
__device__
void
compute_all_Mkernels_v2( const double M[4] ,
			 const double xv[4] ,
			 const double yv[4] ,
			 const struct QED_kernel_temps t ,
			 struct QED_Kernels *K )
{
  // we only need 6 kernels to compute everything
  double Lxy[6][4][4][4] , Lyx[6][4][4][4] , Lxmy[6][4][4][4] ;

  const double myv[4] = { -yv[0] , -yv[1] , -yv[2] , -yv[3] } ;
  QED_kernel_L0( xv , yv  , t , Lxy  ) ;
  QED_kernel_L0( xv , myv , t , Lxmy ) ;
  QED_kernel_L0( yv , xv  , t , Lyx  ) ;

  // and these temporaries: L(x,0) L(y,0) L(0,x) L(0,y)
  double Lx0[6][4][4][4] , L0x[6][4][4][4] ;
  double Ly0[6][4][4][4] , L0y[6][4][4][4] ;
  const double zero[4] = { 0. , 0. , 0. , 0. } ;

  QED_kernel_L0( xv , zero , t , Lx0 ) ;
  QED_kernel_L0( zero , xv , t , L0x ) ;

  QED_kernel_L0( yv , zero , t , Ly0 ) ;
  QED_kernel_L0( zero , yv , t , L0y ) ;

    // precompute x^2, y^2
  const double xsq = xv[0]*xv[0]+xv[1]*xv[1]+xv[2]*xv[2]+xv[3]*xv[3] ;
  const double ysq = yv[0]*yv[0]+yv[1]*yv[1]+yv[2]*yv[2]+yv[3]*yv[3] ;

  // precompute the gaussians
  const double eX[4] = { exp(-M[0]*xsq/2.) , exp(-M[1]*xsq/2.) ,
			 exp(-M[2]*xsq/2.) , exp(-M[3]*xsq/2.) } ;
  const double eY[4] = { exp(-M[0]*ysq/2.) , exp(-M[1]*ysq/2.) ,
			 exp(-M[2]*ysq/2.) , exp(-M[3]*ysq/2.) } ;

  // look-up table precomputations
  const double Mx[4][4] = { { M[0]*xv[0] , M[1]*xv[0] , M[2]*xv[0] , M[3]*xv[0] } ,
			    { M[0]*xv[1] , M[1]*xv[1] , M[2]*xv[1] , M[3]*xv[1] } ,
			    { M[0]*xv[2] , M[1]*xv[2] , M[2]*xv[2] , M[3]*xv[2] } ,
			    { M[0]*xv[3] , M[1]*xv[3] , M[2]*xv[3] , M[3]*xv[3] } } ;
  const double My[4][4] = { { M[0]*yv[0] , M[1]*yv[0] , M[2]*yv[0] , M[3]*yv[0] } ,
			    { M[0]*yv[1] , M[1]*yv[1] , M[2]*yv[1] , M[3]*yv[1] } ,
			    { M[0]*yv[2] , M[1]*yv[2] , M[2]*yv[2] , M[3]*yv[2] } ,
			    { M[0]*yv[3] , M[1]*yv[3] , M[2]*yv[3] , M[3]*yv[3] } } ;

  size_t rhosig , mu , nu , lambda ;
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {
      for( nu = 0 ; nu < 4 ; nu++ ) {
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {
	  // M0
	  K->L0.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(0) +
	    Lyx[rhosig][nu][mu][lambda] - inlineYX2(0) ;
	  // M1
	  K->L1.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(1) +
	    Lyx[rhosig][nu][mu][lambda] - inlineYX2(1) ;
	  // M2
	  K->L2.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(2) +
	    Lyx[rhosig][nu][mu][lambda] - inlineYX2(2) ;
	  // M3
	  K->L3.xy[rhosig][mu][nu][lambda] =
	    Lxy[rhosig][mu][nu][lambda] - inlineXY(3) +
	    Lyx[rhosig][nu][mu][lambda] - inlineYX2(3) ;
	  // and the -y terms
	  K->L0.yx[rhosig][mu][nu][lambda] =
	    Lxmy[rhosig][mu][nu][lambda] - inlineXMY(0) ;
	  K->L1.yx[rhosig][mu][nu][lambda] =
	    Lxmy[rhosig][mu][nu][lambda] - inlineXMY(1) ;
	  K->L2.yx[rhosig][mu][nu][lambda] =
	    Lxmy[rhosig][mu][nu][lambda] - inlineXMY(2) ;
	  K->L3.yx[rhosig][mu][nu][lambda] =
	    Lxmy[rhosig][mu][nu][lambda] - inlineXMY(3) ;
	}
      }
    }
  }
  return ;
}

// swaps L_{..mu,nu..}(y,x) and  L_{..nu,mu..}(y,x) for all kernels
__device__
void
swap_munu_Lyx( struct QED_Kernels *K )
{
  size_t rhosig , mu , nu , lambda ;
  double tmp ;
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {
      for( nu = mu+1 ; nu < 4 ; nu++ ) {
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {
	  // L0 kernel
	  tmp = K -> L0.yx[rhosig][nu][mu][lambda] ;
	  K -> L0.yx[rhosig][nu][mu][lambda] =
	    K -> L0.yx[rhosig][mu][nu][lambda] ;
	  K -> L0.yx[rhosig][mu][nu][lambda] = tmp ;
	  // L1 kernel
	  tmp = K -> L1.yx[rhosig][nu][mu][lambda] ;
	  K -> L1.yx[rhosig][nu][mu][lambda] =
	    K -> L1.yx[rhosig][mu][nu][lambda] ;
	  K -> L1.yx[rhosig][mu][nu][lambda] = tmp ;
	  // L2 kernel
	  tmp = K -> L2.yx[rhosig][nu][mu][lambda] ;
	  K -> L2.yx[rhosig][nu][mu][lambda] =
	    K -> L2.yx[rhosig][mu][nu][lambda] ;
	  K -> L2.yx[rhosig][mu][nu][lambda] = tmp ;
	  // L3 kernel
	  tmp = K -> L3.yx[rhosig][nu][mu][lambda] ;
	  K -> L3.yx[rhosig][nu][mu][lambda] =
	    K -> L3.yx[rhosig][mu][nu][lambda] ;
	  K -> L3.yx[rhosig][mu][nu][lambda] = tmp ;
	}
      }
    }
  }	
  
  return ;
}
