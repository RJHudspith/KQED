#include "hip/hip_runtime.h"
/**
   @file kernels.c
   @brief kernel codes
 */
#include "KQED.h"

#include "kernels.h"        // alphabetising
#include "QED_kernel.h"     // kernelQED()
#include "QED_kernel_xy0.h" // kernelQED_xoryeq0()
#include "Tabd.h"

// check that a 4-vector is zero
__device__
static inline bool
is_zero( const double x[4] )
{
  return (x[0]*x[0]+x[1]*x[1]+x[2]*x[2]+x[3]*x[3]) < 1E-28 ;
}

// check if x vector is equal to y vector
__device__
static inline bool
x_is_y( const double x[4] ,
	const double y[4] )
{
  const double xmy[4] = {x[0]-y[0],x[1]-y[1],x[2]-y[2],x[3]-y[3]} ;
  return is_zero( xmy ) ;
}

// from linear idx in 0->384 return a value where mu and lambda indices are swapped
__device__
static inline size_t
i_to_mulam( const size_t idx )
{
  const size_t l[4] = { idx/64 , (idx/16)&3 , (idx/4)&3 , idx&3 } ;
  return l[1] + 4*(l[2]+4*(l[3]+4*l[0]) ) ;
}

// sets L -> -L_{mu<->lambda}
__device__
static void
kernel_mulam_minus( double *kerv )
{
  double ktmp[384] ;
  memcpy( ktmp , kerv , 384*sizeof(double) ) ;
  size_t i ;
  for( i = 0 ; i < 384 ; i++ ) {
    *kerv = -ktmp[ i_to_mulam(i) ] ;
    kerv++ ;
  }
  return ;
}

// computes kerv += kt * S index-by-index
__device__
static void
atomic_kernel_Saxpy( double *kerv ,
		     const double S ,
		     const double *kt )
{
  int i ;
  // although not strictly necessary is a nice proof that we
  // are correctly aligned to the boundary
  for( i = 0 ; i < 384 ; i++ ) {
    *kerv += *kt * S ;
    kerv++ ; kt++ ;
  }
  return ;
}

// L0 is the standard version of the kernel
__device__
void
QED_kernel_L0( const double xv[4] ,
	       const double yv[4] ,
	       const struct QED_kernel_temps t ,
	       double kerv[6][4][4][4] )
{
  const bool x_is_zero  = is_zero( xv ) ;
  const bool y_is_zero  = is_zero( yv ) ;

  // set kernel to zero
  memset( kerv , 0 , 384*sizeof( double ) ) ;
  
  // do the logic
  if( x_is_zero && y_is_zero ) {
    return ;
  }
  if( x_is_zero ) {
    kernelQED_xoryeq0( yv , t , kerv , Tabd_xeq0 ) ;
    return ;
  }
  if( y_is_zero ) {
    kernelQED_xoryeq0( xv , t , kerv , Tabd_yeq0 ) ;
    return ;
  }
  if( x_is_y( xv , yv ) ) {
    kernelQED_xoryeq0( xv , t , kerv , Tabd_yeq0 ) ;
    kernel_mulam_minus( (double*)kerv ) ;
    return ;
  }
  
  // if neither are zero or small we do the standard one
  kernelQED( xv , yv , t , kerv ) ;
  
  return ;
}

// L1 = L0(x,y) - L0(x,x)/2 - L0(y,y)/2
__device__
void
QED_kernel_L1( const double xv[4] ,
	       const double yv[4] ,
	       const struct QED_kernel_temps t ,
	       double kerv[6][4][4][4] )
{
  double kt[6][4][4][4] KQED_ALIGN ;
  QED_kernel_L0( xv , yv , t , kerv ) ;

  QED_kernel_L0( xv , xv , t , kt ) ;
  atomic_kernel_Saxpy( (double*)kerv , -0.5 , (const double*)kt ) ; 

  QED_kernel_L0( yv , yv , t , kt ) ;
  atomic_kernel_Saxpy( (double*)kerv , -0.5 , (const double*)kt ) ;
  
  return ;
}

// L2 = L0(x,y) - L0(0,y) - L0(x,0)
__device__
void
QED_kernel_L2( const double xv[4] ,
	       const double yv[4] ,
	       const struct QED_kernel_temps t ,
	       double kerv[6][4][4][4] )
{
  double kt[6][4][4][4] KQED_ALIGN ;
  const double zero[4] = { 0 , 0 , 0 , 0 } ;
  QED_kernel_L0( xv , yv , t , kerv ) ;
  
  QED_kernel_L0( zero , yv , t , kt ) ;
  atomic_kernel_Saxpy( (double*)kerv , -1 , (const double*)kt ) ;
  
  QED_kernel_L0( xv , zero , t , kt ) ;
  atomic_kernel_Saxpy( (double*)kerv , -1 , (const double*)kt ) ;
  
  return ;
}

// L3 = L0(x,y) - L0(x,x) + L0(0,x) - L0(0,y)
__device__
void
QED_kernel_L3( const double xv[4] ,
	       const double yv[4] ,
	       const struct QED_kernel_temps t ,
	       double kerv[6][4][4][4] )
{
  double kt[6][4][4][4] KQED_ALIGN ;
  const double zero[4] = { 0 , 0 , 0 , 0 } ;
  QED_kernel_L0( xv , yv , t , kerv ) ;

  QED_kernel_L0( xv , xv , t , kt ) ;
  atomic_kernel_Saxpy( (double*)kerv , -1 , (const double*)kt ) ;

  QED_kernel_L0( zero , xv , t , kt ) ;
  atomic_kernel_Saxpy( (double*)kerv , +1 , (const double*)kt ) ;

  QED_kernel_L0( zero , yv , t , kt ) ;
  atomic_kernel_Saxpy( (double*)kerv , -1 , (const double*)kt ) ;
  
  return ;
}

// L2 = L0(x,y) - d_\mu(x_\alpha exp^{-Mx^2/2}) [L0(0,y) - L0(x,0)]
__device__
void
QED_Mkernel_L2( const double M ,
		const double xv[4] ,
		const double yv[4] ,
		const struct QED_kernel_temps t ,
		double kerv[6][4][4][4] )
{   
  double kt[6][4][4][4] KQED_ALIGN ;
  const double zero[4] = { 0 , 0 , 0 , 0 } ;
  QED_kernel_L0( xv , yv , t , kerv ) ;

  int rhosig , mu , nu , lambda ;
  const double gaussX = exp( -M*(xv[0]*xv[0]+xv[1]*xv[1]+
				 xv[2]*xv[2]+xv[3]*xv[3] )/2. ) ;

  QED_kernel_L0( zero , yv , t , kt ) ;
  
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {
      for( nu = 0 ; nu < 4 ; nu++ ) {
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {
	  kerv[rhosig][mu][nu][lambda] -=
	    gaussX*( kt[rhosig][mu][nu][lambda] -
		     M*xv[mu]*( xv[0]*kt[rhosig][0][nu][lambda] +
				xv[1]*kt[rhosig][1][nu][lambda] +
				xv[2]*kt[rhosig][2][nu][lambda] +
				xv[3]*kt[rhosig][3][nu][lambda] ) ) ;
	}
      }
    }
  }

  const double gaussY = exp( -M*(yv[0]*yv[0]+yv[1]*yv[1]+
				 yv[2]*yv[2]+yv[3]*yv[3] )/2. ) ;
  QED_kernel_L0( xv , zero , t , kt ) ;
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {
      for( nu = 0 ; nu < 4 ; nu++ ) {
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {
	  kerv[rhosig][mu][nu][lambda] -=
	    gaussY*( kt[rhosig][mu][nu][lambda] -
		     M*yv[nu]*( yv[0]*kt[rhosig][mu][0][lambda] +
				yv[1]*kt[rhosig][mu][1][lambda] +
				yv[2]*kt[rhosig][mu][2][lambda] +
				yv[3]*kt[rhosig][mu][3][lambda] ) ) ;
	}
      }
    }
  }
  
  return ;
}
