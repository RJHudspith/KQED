#include "hip/hip_runtime.h"
/**
   @file con_kernel.c
   @brief computes the kernel needed for the connected contribution

   ( dropping rho,sigma arguments)
   
   A = L_{\mu\nu\lambda}(x,y)
   B = L_{\lambda\nu\mu}(x,x-y)
   C = L_{\mu\nu\lambda}(x,y) + L_{\nu\mu\lambda}(y,x) - L_{\lambda\nu\mu}(x,x-y)
*/
#include "KQED.h"

#include "kernels.h"

// computes *K = 0.5*( *K + *W ) for all con kernels in K and W
__device__
void
average_con_kernels( struct Kernels *K ,
		     const struct Kernels *W )
{
  // point out elements of K
  double *KL0A = (double*)K[0].L0 , *KL0B = (double*)K[1].L0 ;
  double *KL0C = (double*)K[2].L0 , *KL1A = (double*)K[0].L1 ;
  double *KL1B = (double*)K[1].L1 , *KL1C = (double*)K[2].L1 ;
  double *KL2A = (double*)K[0].L2 , *KL2B = (double*)K[1].L2 ;
  double *KL2C = (double*)K[2].L2 , *KL3A = (double*)K[0].L3 ;
  double *KL3B = (double*)K[1].L3 , *KL3C = (double*)K[2].L3 ;
  
  // const point out elements of W
  const double *WL0A = (const double*)W[0].L0 ;
  const double *WL1A = (const double*)W[0].L1 ;
  const double *WL2A = (const double*)W[0].L2 ;
  const double *WL3A = (const double*)W[0].L3 ;
  const double *WL3B = (const double*)W[1].L3 ;
  const double *WL0B = (const double*)W[1].L0 ;
  const double *WL1B = (const double*)W[1].L1 ;
  const double *WL2B = (const double*)W[1].L2 ; 
  const double *WL0C = (const double*)W[2].L0 ;
  const double *WL1C = (const double*)W[2].L1 ;
  const double *WL2C = (const double*)W[2].L2 ;
  const double *WL3C = (const double*)W[2].L3 ;
  
  // do the average
  size_t i ;
  for( i = 0 ; i < 384 ; i++ ) {
    *KL0A = 0.5*( *KL0A + *WL0A ) ; KL0A++ ; WL0A++ ;
    *KL0B = 0.5*( *KL0B + *WL0B ) ; KL0B++ ; WL0B++ ;
    *KL0C = 0.5*( *KL0C + *WL0C ) ; KL0C++ ; WL0C++ ;

    *KL1A = 0.5*( *KL1A + *WL1A ) ; KL1A++ ; WL1A++ ;
    *KL1B = 0.5*( *KL1B + *WL1B ) ; KL1B++ ; WL1B++ ;
    *KL1C = 0.5*( *KL1C + *WL1C ) ; KL1C++ ; WL1C++ ;

    *KL2A = 0.5*( *KL2A + *WL2A ) ; KL2A++ ; WL2A++ ;
    *KL2B = 0.5*( *KL2B + *WL2B ) ; KL2B++ ; WL2B++ ;
    *KL2C = 0.5*( *KL2C + *WL2C ) ; KL2C++ ; WL2C++ ;
    
    *KL3A = 0.5*( *KL3A + *WL3A ) ; KL3A++ ; WL3A++ ;
    *KL3B = 0.5*( *KL3B + *WL3B ) ; KL3B++ ; WL3B++ ;
    *KL3C = 0.5*( *KL3C + *WL3C ) ; KL3C++ ; WL3C++ ;
  }
  return ;
}	 

__device__
void
compute_con_kernels_v2( const double xv[4] ,
			const double yv[4] ,
			const struct QED_kernel_temps t ,
			struct Kernels *K )
{
  const double xmyv[4] = { xv[0]-yv[0] , xv[1]-yv[1] ,
			   xv[2]-yv[2] , xv[3]-yv[3] } ;
  const double zero[4] = { 0. , 0. , 0. , 0. } ;
  double Kx_xmy[6][4][4][4] , Ky_x[6][4][4][4] ;
  QED_kernel_L0( xv , yv , t , K[0].L0 ) ;
  QED_kernel_L0( yv , xv , t , Ky_x ) ;
  QED_kernel_L0( xv , xmyv , t , Kx_xmy ) ;

  // and then the ones with a zero
  double Kx_0[6][4][4][4]   , K0_x[6][4][4][4] ;
  double Ky_0[6][4][4][4]   , K0_y[6][4][4][4] ;
  double Kxmy_0[6][4][4][4] , K0_xmy[6][4][4][4] ;
  QED_kernel_L0( xv , zero , t , Kx_0 ) ;
  QED_kernel_L0( zero , xv , t , K0_x ) ;
  QED_kernel_L0( yv , zero , t , Ky_0 ) ;
  QED_kernel_L0( zero , yv , t , K0_y ) ;
  QED_kernel_L0( xmyv , zero , t , Kxmy_0 ) ;
  QED_kernel_L0( zero , xmyv , t , K0_xmy ) ;

  size_t rhosig , mu , nu , lambda ;
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {
      for( nu = 0 ; nu < 4 ; nu++ ) {
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {
	  // L0 kernel
	  // x,x-y
	  K[1].L0[rhosig][mu][nu][lambda] =
	    Kx_xmy[rhosig][lambda][nu][mu] ; 
	  K[2].L0[rhosig][mu][nu][lambda] =
	    K[0].L0[rhosig][mu][nu][lambda]
	      +Ky_x[rhosig][nu][mu][lambda]
	    -Kx_xmy[rhosig][lambda][nu][mu] ;
	  // L1 kernel
	  K[0].L1[rhosig][mu][nu][lambda] = 
	    K[0].L0[rhosig][mu][nu][lambda] +
	    ( +Kx_0[rhosig][lambda][nu][mu]
	      +Ky_0[rhosig][lambda][nu][mu] )/2 ;
	  K[1].L1[rhosig][mu][nu][lambda] = 
	      K[1].L0[rhosig][mu][nu][lambda] +
	    (   +Kx_0[rhosig][mu][nu][lambda]
	      +Kxmy_0[rhosig][mu][nu][lambda] )/2 ;
	  K[2].L1[rhosig][mu][nu][lambda] = 
	    K[0].L1[rhosig][mu][nu][lambda] -
	    K[1].L1[rhosig][mu][nu][lambda] +
	       Ky_x[rhosig][nu][mu][lambda] +
	    ( +Ky_0[rhosig][lambda][mu][nu]
	      +Kx_0[rhosig][lambda][mu][nu] )/2 ;
	  // L2 kernel
	  K[0].L2[rhosig][mu][nu][lambda] = 
	    K[0].L0[rhosig][mu][nu][lambda] -
	    ( +K0_y[rhosig][mu][nu][lambda]
	      +Kx_0[rhosig][mu][nu][lambda] ) ;
	  K[1].L2[rhosig][mu][nu][lambda] = 
	      K[1].L0[rhosig][mu][nu][lambda] -
	    ( +K0_xmy[rhosig][lambda][nu][mu]
	        +Kx_0[rhosig][lambda][nu][mu] ) ;
	  K[2].L2[rhosig][mu][nu][lambda] =
	    K[0].L2[rhosig][mu][nu][lambda] -
	    K[1].L2[rhosig][mu][nu][lambda] +
	       Ky_x[rhosig][nu][mu][lambda] -
	    ( +K0_x[rhosig][nu][mu][lambda]
	      +Ky_0[rhosig][nu][mu][lambda] ) ;
	  // L3 kernel
	  K[0].L3[rhosig][mu][nu][lambda] = 
	    K[0].L0[rhosig][mu][nu][lambda] +
	    ( +Kx_0[rhosig][lambda][nu][mu]
	      +K0_x[rhosig][mu][nu][lambda]
	      -K0_y[rhosig][mu][nu][lambda] ) ;
	  K[1].L3[rhosig][mu][nu][lambda] = 
	      K[1].L0[rhosig][mu][nu][lambda] +
	    (   +Kx_0[rhosig][mu][nu][lambda]
	        +K0_x[rhosig][lambda][nu][mu]
	      -K0_xmy[rhosig][lambda][nu][mu] ) ; 
	  K[2].L3[rhosig][mu][nu][lambda] =
	    K[0].L3[rhosig][mu][nu][lambda] -
	    K[1].L3[rhosig][mu][nu][lambda] +
	       Ky_x[rhosig][nu][mu][lambda] +
	    ( +Ky_0[rhosig][lambda][mu][nu]
	      +K0_y[rhosig][nu][mu][lambda]
	      -K0_x[rhosig][nu][mu][lambda] ) ; 	  
	}
      }
    }
  }
  return ;
}

__device__
void
compute_con_kernels( const double xv[4] ,
		     const double yv[4] ,
		     const struct QED_kernel_temps t ,
		     struct QED_Kernels *K )
{
  const double xmyv[4] = { xv[0]-yv[0] , xv[1]-yv[1] ,
			   xv[2]-yv[2] , xv[3]-yv[3] } ;
  const double zero[4] = { 0. , 0. , 0. , 0. } ;
  double Kx_xmy[6][4][4][4] , Ky_x[6][4][4][4] ;
  QED_kernel_L0( xv , yv , t , K->L0.xy ) ;
  QED_kernel_L0( yv , xv , t , Ky_x ) ;
  QED_kernel_L0( xv , xmyv , t , Kx_xmy ) ;

  // and then the ones with a zero
  double Kx_0[6][4][4][4]   , K0_x[6][4][4][4] ;
  double Ky_0[6][4][4][4]   , K0_y[6][4][4][4] ;
  double Kxmy_0[6][4][4][4] , K0_xmy[6][4][4][4] ;
  QED_kernel_L0( xv , zero , t , Kx_0 ) ;
  QED_kernel_L0( zero , xv , t , K0_x ) ;
  QED_kernel_L0( yv , zero , t , Ky_0 ) ;
  QED_kernel_L0( zero , yv , t , K0_y ) ;
  QED_kernel_L0( xmyv , zero , t , Kxmy_0 ) ;
  QED_kernel_L0( zero , xmyv , t , K0_xmy ) ;

  size_t rhosig , mu , nu , lambda ;
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {
      for( nu = 0 ; nu < 4 ; nu++ ) {
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {
	  // L0 kernel
	  K->L0.xy[rhosig][mu][nu][lambda] =
	    ( +K->L0.xy[rhosig][mu][nu][lambda]
	      +Ky_x[rhosig][nu][mu][lambda]
	      -Kx_xmy[rhosig][lambda][nu][mu] ) ;
	  K->L0.yx[rhosig][mu][nu][lambda] =
	    Kx_xmy[rhosig][lambda][nu][mu] ; 

	  // L1 kernel
	  K->L1.xy[rhosig][mu][nu][lambda] =
	    K->L0.xy[rhosig][mu][nu][lambda] +
	    ( +Kx_0[rhosig][lambda][nu][mu]
	      +Ky_0[rhosig][lambda][nu][mu]
	      +Kx_0[rhosig][lambda][mu][nu]
	      +Ky_0[rhosig][lambda][mu][nu]
	      -Kx_0[rhosig][mu][nu][lambda]
	      -Kxmy_0[rhosig][mu][nu][lambda] )/2 ;
	  K->L1.yx[rhosig][mu][nu][lambda] = 
	    K->L0.yx[rhosig][mu][nu][lambda] +
	    ( +Kx_0[rhosig][mu][nu][lambda]
	      +Kxmy_0[rhosig][mu][nu][lambda] )/2 ;
	  
	  // L2 kernel
	  K->L2.xy[rhosig][mu][nu][lambda] =
	    K->L0.xy[rhosig][mu][nu][lambda] -
	    ( +K0_y[rhosig][mu][nu][lambda]
	      +Kx_0[rhosig][mu][nu][lambda]
	      +K0_x[rhosig][nu][mu][lambda]
	      +Ky_0[rhosig][nu][mu][lambda]
	      -K0_xmy[rhosig][lambda][nu][mu]
	      -Kx_0[rhosig][lambda][nu][mu] ) ;
	  K->L2.yx[rhosig][mu][nu][lambda] = 
	    K->L0.yx[rhosig][mu][nu][lambda] -
	    ( +Kx_0[rhosig][lambda][nu][mu]
	      +K0_xmy[rhosig][lambda][nu][mu] ) ;

	  // L3 kernel
	  K->L3.xy[rhosig][mu][nu][lambda] =
	    K->L2.xy[rhosig][mu][nu][lambda] +
	    ( K0_x[rhosig][mu][nu][lambda] +
	      K0_y[rhosig][nu][mu][lambda] +
	      Ky_0[rhosig][lambda][mu][nu] -
	      K0_x[rhosig][lambda][nu][mu] +
	      Ky_0[rhosig][nu][mu][lambda] ) ;
	  K->L3.yx[rhosig][mu][nu][lambda] = 
	    K->L0.yx[rhosig][mu][nu][lambda] +
	    ( +Kx_0[rhosig][mu][nu][lambda]
	      +K0_x[rhosig][lambda][nu][mu]
	      -K0_xmy[rhosig][lambda][nu][mu] ) ;
	}
      }
    }
  }
	  
  return ;
}

// hmmm apparently this is pretty slow, perhaps I need to do something
// drastic about it
#define UNROLL_NM(Nm)							\
  LMxy[Nm] = Kxy[rhosig][mu][nu][lambda]				\
    -*eXP*( K0y[rhosig][mu][nu][lambda] - Mx[mu][Nm]*Pf1 )		\
    -*eYP*( Kx0[rhosig][mu][nu][lambda] - My[nu][Nm]*Pf2[lambda] )  ;	\
  LMyx[Nm] = Kyx[rhosig][nu][mu][lambda]				\
    -*eYP*( K0x[rhosig][nu][mu][lambda]-My[nu][Nm]*Pf3[lambda] )	\
    -*eXP*( Ky0[rhosig][nu][mu][lambda]-Mx[mu][Nm]*Pf4 )  ;		\
  LMxxmy[Nm] = Kxxmy[rhosig][lambda][nu][mu]				\
    -*eXP*( K0xmy[rhosig][lambda][nu][mu]-Mx[lambda][Nm]*Pf5 )		\
    -*eXMYP*( Kx0[rhosig][lambda][nu][mu]-Mxmy[nu][Nm]*Pf6[lambda] )  ; \
  eXP++ ; eYP++ ; eXMYP++ ;

// compute 4 L2 kernels with the M-factor included
__device__
void
compute_con_kernelsM_L2( const double M[4] ,
			 const double xv[4] ,
			 const double yv[4] ,
			 const struct QED_kernel_temps t ,
			 struct Kernels *K )
{
  const double xmyv[4] = { xv[0]-yv[0] , xv[1]-yv[1] , xv[2]-yv[2] , xv[3]-yv[3] } ;
  
  double Kxy[6][4][4][4] KQED_ALIGN , Kxxmy[6][4][4][4] KQED_ALIGN ;
  double Kyx[6][4][4][4] KQED_ALIGN ;
  QED_kernel_L0( xv , yv   , t , Kxy ) ;
  QED_kernel_L0( yv , xv   , t , Kyx ) ;
  QED_kernel_L0( xv , xmyv , t , Kxxmy ) ;

  double Kx0[6][4][4][4] KQED_ALIGN , Ky0[6][4][4][4] KQED_ALIGN ;
  double K0y[6][4][4][4] KQED_ALIGN , K0x[6][4][4][4] KQED_ALIGN ;
  double K0xmy[6][4][4][4] KQED_ALIGN ;
  const double zero[4] = { 0. , 0. , 0. , 0. } ;
  QED_kernel_L0( xv , zero   , t , Kx0   ) ;
  QED_kernel_L0( zero , xv   , t , K0x   ) ;
  QED_kernel_L0( yv , zero   , t , Ky0   ) ;
  QED_kernel_L0( zero , yv   , t , K0y   ) ;
  QED_kernel_L0( zero , xmyv , t , K0xmy ) ;

  // precompute x^2, y^2, (x-y)^2
  const double xsq = xv[0]*xv[0]+xv[1]*xv[1]+xv[2]*xv[2]+xv[3]*xv[3] ;
  const double ysq = yv[0]*yv[0]+yv[1]*yv[1]+yv[2]*yv[2]+yv[3]*yv[3] ;
  const double xmysq = xmyv[0]*xmyv[0]+xmyv[1]*xmyv[1]+xmyv[2]*xmyv[2]+xmyv[3]*xmyv[3] ;

  // precompute the gaussians
  const double eX[4] KQED_ALIGN = { exp(-M[0]*xsq/2.) , exp(-M[1]*xsq/2.) ,
				    exp(-M[2]*xsq/2.) , exp(-M[3]*xsq/2.) } ;
  const double eY[4] KQED_ALIGN = { exp(-M[0]*ysq/2.) , exp(-M[1]*ysq/2.) ,
				    exp(-M[2]*ysq/2.) , exp(-M[3]*ysq/2.) } ;
  const double eXMY[4] KQED_ALIGN = { exp(-M[0]*xmysq/2.) , exp(-M[1]*xmysq/2.) ,
				      exp(-M[2]*xmysq/2.) , exp(-M[3]*xmysq/2.) } ;

  // look-up table precomputations
  const double Mx[4][4] KQED_ALIGN = { { M[0]*xv[0] , M[1]*xv[0] , M[2]*xv[0] , M[3]*xv[0] } ,
			    { M[0]*xv[1] , M[1]*xv[1] , M[2]*xv[1] , M[3]*xv[1] } ,
			    { M[0]*xv[2] , M[1]*xv[2] , M[2]*xv[2] , M[3]*xv[2] } ,
			    { M[0]*xv[3] , M[1]*xv[3] , M[2]*xv[3] , M[3]*xv[3] } } ;
  const double My[4][4] KQED_ALIGN = { { M[0]*yv[0] , M[1]*yv[0] , M[2]*yv[0] , M[3]*yv[0] } ,
			    { M[0]*yv[1] , M[1]*yv[1] , M[2]*yv[1] , M[3]*yv[1] } ,
			    { M[0]*yv[2] , M[1]*yv[2] , M[2]*yv[2] , M[3]*yv[2] } ,
			    { M[0]*yv[3] , M[1]*yv[3] , M[2]*yv[3] , M[3]*yv[3] } } ;
  const double Mxmy[4][4] KQED_ALIGN = { { M[0]*xmyv[0] , M[1]*xmyv[0] , M[2]*xmyv[0] , M[3]*xmyv[0] } ,
			      { M[0]*xmyv[1] , M[1]*xmyv[1] , M[2]*xmyv[1] , M[3]*xmyv[1] } ,
			      { M[0]*xmyv[2] , M[1]*xmyv[2] , M[2]*xmyv[2] , M[3]*xmyv[2] } ,
			      { M[0]*xmyv[3] , M[1]*xmyv[3] , M[2]*xmyv[3] , M[3]*xmyv[3] } } ;

  // calculate the kernels
  int rhosig , mu , nu , lambda ;
  for( rhosig = 0 ; rhosig < 6 ; rhosig++ ) {
    for( mu = 0 ; mu < 4 ; mu++ ) {

      // precomputations where I have pulled out the lambda dependence
      // and so these only depend on rhosig and mu indices
      const double Pf2[4] KQED_ALIGN =
	{ yv[0]*Kx0[rhosig][mu][0][0] + yv[1]*Kx0[rhosig][mu][1][0] +
	  yv[2]*Kx0[rhosig][mu][2][0] + yv[3]*Kx0[rhosig][mu][3][0] ,
	  yv[0]*Kx0[rhosig][mu][0][1] + yv[1]*Kx0[rhosig][mu][1][1] +
	  yv[2]*Kx0[rhosig][mu][2][1] + yv[3]*Kx0[rhosig][mu][3][1] ,
	  yv[0]*Kx0[rhosig][mu][0][2] + yv[1]*Kx0[rhosig][mu][1][2] +
	  yv[2]*Kx0[rhosig][mu][2][2] + yv[3]*Kx0[rhosig][mu][3][2] ,
	  yv[0]*Kx0[rhosig][mu][0][3] + yv[1]*Kx0[rhosig][mu][1][3] +
	  yv[2]*Kx0[rhosig][mu][2][3] + yv[3]*Kx0[rhosig][mu][3][3] } ;
      const double Pf3[4] KQED_ALIGN =
	{ yv[0]*K0x[rhosig][0][mu][0] + yv[1]*K0x[rhosig][1][mu][0] +
	  yv[2]*K0x[rhosig][2][mu][0] + yv[3]*K0x[rhosig][3][mu][0] ,
	  yv[0]*K0x[rhosig][0][mu][1] + yv[1]*K0x[rhosig][1][mu][1] +
	  yv[2]*K0x[rhosig][2][mu][1] + yv[3]*K0x[rhosig][3][mu][1] ,
	  yv[0]*K0x[rhosig][0][mu][2] + yv[1]*K0x[rhosig][1][mu][2] +
	  yv[2]*K0x[rhosig][2][mu][2] + yv[3]*K0x[rhosig][3][mu][2] ,
	  yv[0]*K0x[rhosig][0][mu][3] + yv[1]*K0x[rhosig][1][mu][3] +
	  yv[2]*K0x[rhosig][2][mu][3] + yv[3]*K0x[rhosig][3][mu][3] } ;
      const double Pf6[4] KQED_ALIGN =
	{ xmyv[0]*Kx0[rhosig][0][0][mu] + xmyv[1]*Kx0[rhosig][0][1][mu] +
	  xmyv[2]*Kx0[rhosig][0][2][mu] + xmyv[3]*Kx0[rhosig][0][3][mu] ,
	  xmyv[0]*Kx0[rhosig][1][0][mu] + xmyv[1]*Kx0[rhosig][1][1][mu] +
	  xmyv[2]*Kx0[rhosig][1][2][mu] + xmyv[3]*Kx0[rhosig][1][3][mu] ,
	  xmyv[0]*Kx0[rhosig][2][0][mu] + xmyv[1]*Kx0[rhosig][2][1][mu] +
	  xmyv[2]*Kx0[rhosig][2][2][mu] + xmyv[3]*Kx0[rhosig][2][3][mu] ,
	  xmyv[0]*Kx0[rhosig][3][0][mu] + xmyv[1]*Kx0[rhosig][3][1][mu] +
	  xmyv[2]*Kx0[rhosig][3][2][mu] + xmyv[3]*Kx0[rhosig][3][3][mu] } ;
      
      for( nu = 0 ; nu < 4 ; nu++ ) {

	double Pf5 =
	  xv[0]*K0xmy[rhosig][0][nu][mu] +
	  xv[1]*K0xmy[rhosig][1][nu][mu] +
	  xv[2]*K0xmy[rhosig][2][nu][mu] +
	  xv[3]*K0xmy[rhosig][3][nu][mu] ;
	
	for( lambda = 0 ; lambda < 4 ; lambda++ ) {

	  // temporary storage
	  double LMxy[4] KQED_ALIGN , LMyx[4] KQED_ALIGN , LMxxmy[4] KQED_ALIGN ;

	  // inner products go here
	  const double Pf1 =
	    xv[0]*K0y[rhosig][0][nu][lambda] +
	    xv[1]*K0y[rhosig][1][nu][lambda] +
	    xv[2]*K0y[rhosig][2][nu][lambda] +
	    xv[3]*K0y[rhosig][3][nu][lambda] ;
	    
	  const double Pf4 =
	    xv[0]*Ky0[rhosig][nu][0][lambda] +
	    xv[1]*Ky0[rhosig][nu][1][lambda] +
	    xv[2]*Ky0[rhosig][nu][2][lambda] +
	    xv[3]*Ky0[rhosig][nu][3][lambda] ;

	  const double *eXP   = (const double*)eX ;
	  const double *eYP   = (const double*)eY ;
	  const double *eXMYP = (const double*)eXMY ;
	  UNROLL_NM(0);
	  UNROLL_NM(1);
	  UNROLL_NM(2);
	  UNROLL_NM(3); 
	  
	  K[0].L0[rhosig][mu][nu][lambda] = LMxy[0] ;
	  K[1].L0[rhosig][mu][nu][lambda] = LMxxmy[0] ;
	  K[2].L0[rhosig][mu][nu][lambda] = LMxy[0]+LMyx[0]-LMxxmy[0] ;  

	  K[0].L1[rhosig][mu][nu][lambda] = LMxy[1] ;
	  K[1].L1[rhosig][mu][nu][lambda] = LMxxmy[1] ;
	  K[2].L1[rhosig][mu][nu][lambda] = LMxy[1]+LMyx[1]-LMxxmy[1] ;  
	  
	  K[0].L2[rhosig][mu][nu][lambda] = LMxy[2] ;
	  K[1].L2[rhosig][mu][nu][lambda] = LMxxmy[2] ;
	  K[2].L2[rhosig][mu][nu][lambda] = LMxy[2]+LMyx[2]-LMxxmy[2] ;  
	  
	  K[0].L3[rhosig][mu][nu][lambda] = LMxy[3] ;
	  K[1].L3[rhosig][mu][nu][lambda] = LMxxmy[3] ;
	  K[2].L3[rhosig][mu][nu][lambda] = LMxy[3]+LMyx[3]-LMxxmy[3] ;
	}
      }
    }
  }

  return ;
}
